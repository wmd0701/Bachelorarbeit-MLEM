#include "hip/hip_runtime.h"
#include "algorithm"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "math.h"
#include "kernel.cuh"
#include "hipsparse.h"
#include "csr4matrix.hpp"
#include "vector.hpp"
#include "time.h"
#include "sptrans.h"
#include "nccl.h"

// #define TransposeMatrixUsingCPU true
#define Iterations 300

// 0: naive mlem
// 1: mlem using nccl
#define MLEM_Version 1
#define Naive_Device 0

void csr_format_for_cuda(const Csr4Matrix& matrix, float* csrVal, int* csrRowInd, int* csrColInd){   
    int index = 0;
    csrRowInd[index] = 0;
    // !!! using openMP here will 100% lead to error in matrix
    // #pragma omp parallel for schedule (static)
    for (int row = 0; row < matrix.rows(); ++row) {
        csrRowInd[row + 1] = csrRowInd[row] + (int)matrix.elementsInRow(row);
	
        std::for_each(matrix.beginRow2(row), matrix.endRow2(row),[&](const RowElement<float>& e){ 
            csrVal[index] = e.value();
            csrColInd[index] = (int)e.column() ;
            index = index + 1; }
        );
    }
}

void calcColumnSums(const Csr4Matrix& matrix, Vector<float>& norm)
{
    assert(matrix.columns() == norm.size());

    std::fill(norm.ptr(), norm.ptr() + norm.size(), 0.0);
    matrix.mapRows(0, matrix.rows());

  // pragma omp parallel for schedule (static)
    for (uint32_t row=0; row<matrix.rows(); ++row) {
        std::for_each(matrix.beginRow2(row), matrix.endRow2(row),
                      [&](const RowElement<float>& e){ norm[e.column()] += e.value(); });
    }
    // norm.writeToFile("norm-0.out");
}

void transposeCSR(int *cuda_Rows, int *cuda_Cols, float *cuda_Vals, int *cuda_Rows_Trans, int *cuda_Cols_Trans, float *cuda_Vals_Trans,
                    int rows, int cols, int nnzs){
    hipsparseStatus_t status;
	hipsparseHandle_t handle = 0;
	status = hipsparseCreate(&handle);
	if (status != HIPSPARSE_STATUS_SUCCESS){
        hipError_t cuda_err = hipGetLastError();
        printf("    Fail : CSR to CSC, cusparese initialization failed , ERROR %d, %s\n", status, hipGetErrorString(cuda_err));
    }
    status = hipsparseScsr2csc(handle, rows, cols, nnzs, cuda_Vals, cuda_Rows, cuda_Cols, cuda_Vals_Trans, cuda_Cols_Trans, cuda_Rows_Trans, 
                                HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
    if (status != HIPSPARSE_STATUS_SUCCESS)
        printf("    Fail : CSR to CSC, cusparse transpose failed\n");

    status = hipsparseDestroy(handle);
    handle = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS)
        printf("    Fail : CSR to CSC, cusparse destroy failed\n");
    
    // cusparse functions are asynchronous
    hipDeviceSynchronize();
}


// return row index for in which row the nnzs are distributed into two pars equally
int halfMatrix(int *csr_Rows, int nnzs, int rows){
    int i = 0;
    int halfnnzs = nnzs / 2;
    for(; i <= rows; i++)
        if(csr_Rows[i] >= halfnnzs)
            break;
    return i;
}

/* a general version of halfMatrix: partition matrix into device_numbers parts, corresponding rows are saved in the array segments
   start row of segment i: segments[i]
    end  row of segment i: segments[i+1]
    number of rows in segment i: segments[i+1] - segments[i] (saved in segment_rows)
    number of nnzs in segment i: csr_Rows[segments[i+1]] - csr_Rows[segments[i]] (saved in segment_nnzs)
    offset when copying from host to device: csr_Rows[segments[i]] (saved in offsets)
*/
void partitionMatrix(int *csr_Rows, int nnzs, int rows, int device_numbers, int *segments, int *segment_rows, int *segment_nnzs, int *offsets){
    segments[0] = 0;
    segments[device_numbers] = rows;
    int i = 0;
    int nnzs_per_segment = nnzs / device_numbers;
    for(int segment = 1; segment < device_numbers; segment++){
        for(; i <= rows; i++)
            if(csr_Rows[i] >= nnzs_per_segment * segment)
                break;
        segments[segment] = i;
    }
    for(int segment = 0; segment < device_numbers; segment++){
        segment_rows[segment] = segments[segment+1] - segments[segment];
        segment_nnzs[segment] = csr_Rows[segments[segment+1]] - csr_Rows[segments[segment]];
        offsets[segment] = csr_Rows[segments[segment]];
    }
}

void mlem_nccl( int *csr_Rows, int *csr_Cols, float *csr_Vals,
                int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, 
                int *g, float *norm, float *f, int rows, int cols, int nnzs){
    
    int device_numbers;
    hipGetDeviceCount(&device_numbers);
    if(device_numbers < 2){
        printf("    \nWarning! Number of capable GPUs less than 2!\n\n");
        return;
    }
    else
        printf("    \nRunning NCCL MLEM with %d CUDA devices\n\n", device_numbers);

    clock_t start = clock();
    printf("    Begin: Initialization\n");
    clock_t initStart = clock();

    // partition matrix
    int *segments = (int*)malloc((device_numbers+1)*sizeof(int));
    int *segment_rows = (int*)malloc(device_numbers*sizeof(int));
    int *segment_nnzs = (int*)malloc(device_numbers*sizeof(int));
    int *offsets = (int*)malloc(device_numbers*sizeof(int));
    partitionMatrix(csr_Rows, nnzs, rows, device_numbers, segments, segment_rows, segment_nnzs, offsets);


    // partition transposed matrix
    int *segments_trans = (int*)malloc((device_numbers+1)*sizeof(int));
    int *segment_rows_trans = (int*)malloc(device_numbers*sizeof(int));
    int *segment_nnzs_trans = (int*)malloc(device_numbers*sizeof(int));
    int *offsets_trans = (int*)malloc(device_numbers*sizeof(int));
    partitionMatrix(csr_Rows_Trans, nnzs, cols, device_numbers, segments_trans, segment_rows_trans, segment_nnzs_trans, offsets_trans);
    
    
    // NCCL elements
    ncclComm_t *comms = (ncclComm_t*)malloc(device_numbers * sizeof(ncclComm_t));;
    hipStream_t *streams = (hipStream_t*)malloc(device_numbers * sizeof(hipStream_t));
    int *devices = (int*)malloc(device_numbers * sizeof(int));    


    // device variables
    int **cuda_Rows = (int**)malloc(device_numbers*sizeof(int*));
    int **cuda_Cols = (int**)malloc(device_numbers*sizeof(int*)); 
    int **cuda_Rows_Trans = (int**)malloc(device_numbers*sizeof(int*));
    int **cuda_Cols_Trans = (int**)malloc(device_numbers*sizeof(int*));
    int **cuda_g = (int**)malloc(device_numbers*sizeof(int*));
    float **cuda_Vals = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_Vals_Trans = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_norm = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_bwproj = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_temp = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_f = (float**)malloc(device_numbers*sizeof(float*));


    // initialization
    int blocksize = 1024;   // unique blocksize for all kernel calls
    int *gridsize_fwproj = (int*)malloc(device_numbers*sizeof(int));
    int *gridsize_correl = (int*)malloc(device_numbers*sizeof(int));
    int *gridsize_bwproj = (int*)malloc(device_numbers*sizeof(int));
    int *gridsize_update = (int*)malloc(device_numbers*sizeof(int));
    int *secsize_fwproj = (int*)malloc(device_numbers*sizeof(int));
    int *secsize_bwproj = (int*)malloc(device_numbers*sizeof(int));
    for(int i = 0; i < device_numbers; i++){
        hipSetDevice(i);
        hipStreamCreate(streams+i);
        devices[i] = i;

        hipMalloc((void**)&cuda_Rows[i], sizeof(int)*(segment_rows[i] + 1));
        hipMalloc((void**)&cuda_Cols[i], sizeof(int)*segment_nnzs[i]);
        hipMalloc((void**)&cuda_Vals[i], sizeof(float)*segment_nnzs[i]);
        hipMalloc((void**)&cuda_Rows_Trans[i], sizeof(int)*(segment_rows_trans[i] + 1));
        hipMalloc((void**)&cuda_Cols_Trans[i], sizeof(int)*segment_nnzs_trans[i]);
        hipMalloc((void**)&cuda_Vals_Trans[i], sizeof(float)*segment_nnzs_trans[i]);
        hipMalloc((void**)&cuda_f[i], sizeof(float)*cols);
        hipMalloc((void**)&cuda_bwproj[i], sizeof(float)*cols);
        hipMalloc((void**)&cuda_temp[i], sizeof(float)*rows);
        hipMalloc((void**)&cuda_g[i], sizeof(int)*segment_rows[i]);
        hipMalloc((void**)&cuda_norm[i], sizeof(float)*segment_rows_trans[i]);

        
        // copy matrix from host to devices
        for(int j = segments[i]; j <= segments[i+1]; j++ )
            csr_Rows[j] -= offsets[i];
        hipMemcpy(cuda_Rows[i], csr_Rows+segments[i], sizeof(int)*(segment_rows[i] + 1), hipMemcpyHostToDevice);
        csr_Rows[segments[i+1]] += offsets[i];
        hipMemcpy(cuda_Cols[i], csr_Cols+offsets[i], sizeof(int)*segment_nnzs[i], hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals[i], csr_Vals+offsets[i], sizeof(float)*segment_nnzs[i], hipMemcpyHostToDevice);
        
        // copy transposed matrix from host to devices
        for(int j = segments_trans[i]; j <= segments_trans[i+1]; j++ )
            csr_Rows_Trans[j] -= offsets_trans[i];
        hipMemcpy(cuda_Rows_Trans[i], csr_Rows_Trans+segments_trans[i], sizeof(int)*(segment_rows_trans[i] + 1), hipMemcpyHostToDevice);
        csr_Rows_Trans[segments_trans[i+1]] += offsets_trans[i];
        hipMemcpy(cuda_Cols_Trans[i], csr_Cols_Trans+offsets_trans[i], sizeof(int)*segment_nnzs_trans[i], hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals_Trans[i], csr_Vals_Trans+offsets_trans[i], sizeof(float)*segment_nnzs_trans[i], hipMemcpyHostToDevice);
        
        // copy other vectors from host to devices
        hipMemcpy(cuda_g[i], g+segments[i], sizeof(int)*segment_rows[i], hipMemcpyHostToDevice);
        hipMemcpy(cuda_norm[i], norm+segments_trans[i], sizeof(float)*segment_rows_trans[i], hipMemcpyHostToDevice);
        hipMemset(cuda_bwproj[i], 0, sizeof(float)*cols);
        hipMemset(cuda_temp[i], 0, sizeof(float)*rows);
        hipMemcpy(cuda_f[i], f, sizeof(float)*cols, hipMemcpyHostToDevice);
        
        // determine grid size for each step when calling CUDA kernels
        gridsize_correl[i] = ceil((double)segment_rows[i] / blocksize);
        gridsize_update[i] = ceil((double)segment_rows_trans[i] / blocksize);
        int items_fwproj = segment_rows[i] + segment_nnzs[i];
        int items_bwproj = segment_rows_trans[i] + segment_nnzs_trans[i];
        gridsize_fwproj[i] = ceil(sqrt((double)items_fwproj / blocksize));
        gridsize_bwproj[i] = ceil(sqrt((double)items_bwproj / blocksize));
        // determine section size for foward projection and backward projection
        secsize_fwproj[i] = ceil((double)items_fwproj / (blocksize * gridsize_fwproj[i]));
        secsize_bwproj[i] = ceil((double)items_bwproj / (blocksize * gridsize_bwproj[i]));
    }

    // NCCL initialization
    ncclCommInitAll(comms, device_numbers, devices);
    
    clock_t initEnd = clock();
    printf("    End  : Initialization\n");
    double initTime = ((double) (initEnd - initStart)) / CLOCKS_PER_SEC;
    printf("    Elapsed time for initialization: %f\n\n", initTime);


    // iterations
    printf("    Begin: Iterations %d\n", Iterations);
    clock_t iterStart = clock();
    for(int iter = 0; iter < Iterations; iter++){
        
        // forward projection
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            calcFwProj <<< gridsize_fwproj[i], blocksize >>> (  cuda_Rows[i], cuda_Vals[i], cuda_Cols[i], cuda_f[i], 
                                                                cuda_temp[i] + segments[i], secsize_fwproj[i], segment_rows[i], segment_nnzs[i]);
        }

        // correlation
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            calcCorrel <<< gridsize_correl[i], blocksize >>> (cuda_g[i], cuda_temp[i]+segments[i], segment_rows[i]);
        }

        // sum up cuda_temp over devices
        for (int i = 0; i < device_numbers; ++i) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }
        ncclGroupStart();
        for (int i = 0; i < device_numbers; i++)
            ncclAllReduce((const void*)cuda_temp[i], (void*)cuda_temp[i], rows, ncclFloat, ncclSum, comms[i], streams[i]);
        ncclGroupEnd();
        for (int i = 0; i < device_numbers; ++i) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }

        // backward projection
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            calcBwProj <<< gridsize_bwproj[i], blocksize >>> (  cuda_Rows_Trans[i], cuda_Vals_Trans[i], cuda_Cols_Trans[i], cuda_temp[i], 
                                                                cuda_bwproj[i] + segments_trans[i], secsize_bwproj[i], segment_rows_trans[i], segment_nnzs_trans[i]);
        }

        // update, for mlem nccl calcUpdate should be used, followd by clearing bwproj using cudamemset
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            calcUpdate <<< gridsize_update[i], blocksize >>> (cuda_f[i] + segments_trans[i], cuda_norm[i], cuda_bwproj[i] + segments_trans[i], segment_rows_trans[i]);
        }

        // sum up cuda_bwproj over devices and save in cuda_f
        for (int i = 0; i < device_numbers; ++i) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }
        ncclGroupStart();
        for (int i = 0; i < device_numbers; i++)
            ncclAllReduce((const void*)cuda_bwproj[i], (void*)cuda_f[i], cols, ncclFloat, ncclSum, comms[i], streams[i]);
        ncclGroupEnd();
        for (int i = 0; i < device_numbers; ++i) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }

        // clear cuda_bwproj
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            hipMemset(cuda_bwproj[i], 0, sizeof(float)*cols);
        }

        // clear cuda_temp
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            hipMemset(cuda_temp[i], 0, sizeof(float)*rows);
        }
    }
    clock_t iterEnd = clock();
    printf("    End  : Iterations %d\n", Iterations);
    double itertime = ((double) (iterEnd - iterStart)) / CLOCKS_PER_SEC;
    printf("    Elapsed time for iterations: %f\n\n", itertime);


    // Result is copied to f from device 0, actually now all devices hold the same result
    hipSetDevice(0);
    hipMemcpy(f, cuda_f[0], sizeof(float)*cols, hipMemcpyDeviceToHost);

    // free all memory
    for(int i = 0; i < device_numbers; i++){
        hipSetDevice(i);
        ncclCommDestroy(comms[i]);
        if(cuda_Rows[i]) hipFree(cuda_Rows[i]);
        if(cuda_Cols[i]) hipFree(cuda_Cols[i]);
        if(cuda_Rows_Trans[i]) hipFree(cuda_Rows_Trans[i]);
        if(cuda_Cols_Trans[i]) hipFree(cuda_Cols_Trans[i]);
        if(cuda_g[i]) hipFree(cuda_g[i]);
        if(cuda_Vals[i]) hipFree(cuda_Vals[i]);
        if(cuda_Vals_Trans[i]) hipFree(cuda_Vals_Trans[i]);
        if(cuda_norm[i]) hipFree(cuda_norm[i]);
        if(cuda_bwproj[i]) hipFree(cuda_bwproj[i]);
        if(cuda_temp[i]) hipFree(cuda_temp[i]);
        if(cuda_f[i]) hipFree(cuda_f[i]);
    }
    if(segments) free(segments);
    if(segment_rows) free(segment_rows);
    if(segment_nnzs) free(segment_nnzs);
    if(offsets) free(offsets);
    if(segments_trans) free(segments_trans);
    if(segment_rows_trans) free(segment_rows_trans);
    if(segment_nnzs_trans) free(segment_nnzs_trans);
    if(offsets_trans) free(offsets_trans);
    if(comms) free(comms);
    if(streams) free(streams);
    if(devices) free(devices);
    if(cuda_Rows) free(cuda_Rows);
    if(cuda_Cols) free(cuda_Cols);
    if(cuda_Rows_Trans) free(cuda_Rows_Trans);
    if(cuda_Cols_Trans) free(cuda_Cols_Trans);
    if(cuda_g) free(cuda_g);
    if(cuda_Vals) free(cuda_Vals);
    if(cuda_Vals_Trans) free(cuda_Vals_Trans);
    if(cuda_norm) free(cuda_norm);
    if(cuda_bwproj) free(cuda_bwproj);
    if(cuda_temp) free(cuda_temp);
    if(cuda_f) free(cuda_f);
    if(gridsize_fwproj) free(gridsize_fwproj);
    if(gridsize_correl) free(gridsize_correl);
    if(gridsize_bwproj) free(gridsize_bwproj);
    if(gridsize_update) free(gridsize_update);
    if(secsize_fwproj) free(secsize_fwproj);
    if(secsize_bwproj) free(secsize_bwproj);
    

    clock_t end = clock();
    double totaltime = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("    Elapsed time totally       : %f\n\n", totaltime);
}


void mlem_naive(    int *csr_Rows, int *csr_Cols, float *csr_Vals, 
                    int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, 
                    int *g, float *norm, float *f, int rows, int cols, int nnzs){
    
    // 1: P6000
    hipSetDevice(Naive_Device);
    printf("    \nRunning naive MLEM on CUDA device %d (P6000)\n\n", Naive_Device);
    
    clock_t start = clock();
    printf("    Begin: Initialization\n");
    clock_t initStart = clock();

    // halve the matrix
    int halfrows1 = halfMatrix(csr_Rows, nnzs, rows);
    int halfrows2 = rows - halfrows1;
    int halfnnzs1 = csr_Rows[halfrows1];
    int halfnnzs2 = nnzs - halfnnzs1;
    int offset = csr_Rows[halfrows1];
    // printf("First  half matrix contains %d rows and %d nnzs\n", halfrows1, halfnnzs1);
    // printf("Second half matrix contains %d rows and %d nnzs\n", halfrows2, halfnnzs2);
    // adjust row array for the second half matrix
    // TODO: accelerate this adjustment with GPU
    for(int i = halfrows1+1; i <= rows; i++)
        csr_Rows[i] -= offset;

    // halve the transposed matrix
    int halfrows1_trans = halfMatrix(csr_Rows_Trans, nnzs, cols);
    int halfrows2_trans = cols - halfrows1_trans;
    int halfnnzs1_trans = csr_Rows_Trans[halfrows1_trans];
    int halfnnzs2_trans = nnzs - halfnnzs1_trans;
    int offset_trans = csr_Rows_Trans[halfrows1_trans];
    // printf("First  half transposed matrix contains %d rows and %d nnzs\n", halfrows1_trans, halfnnzs1_trans);
    // printf("Second half transposed matrix contains %d rows and %d nnzs\n", halfrows2_trans, halfnnzs2_trans);
    // adjust row array for the second half matrix
    // TODO: accelerate this adjustment with GPU
    for(int i = halfrows1_trans+1; i <= cols; i++)
        csr_Rows_Trans[i] -= offset_trans;


    // device variables
    int *cuda_Rows, *cuda_Cols, *cuda_Rows_Trans, *cuda_Cols_Trans, *cuda_g;
    float *cuda_Vals, *cuda_Vals_Trans, *cuda_norm, *cuda_bwproj, *cuda_temp, *cuda_f;


    // allocate device storage
    // printf("    Begin: Allocate GPU Storage\n");
    int rows_init = halfrows1 > halfrows2 ? halfrows1 : halfrows2;
    int nnzs_init = halfnnzs1 > halfnnzs2 ? halfnnzs1 : halfnnzs2;
    int rows_init_trans = halfrows1_trans > halfrows2_trans ? halfrows1_trans : halfrows2_trans;
    int nnzs_init_trans = halfnnzs1_trans > halfnnzs2_trans ? halfnnzs1_trans : halfnnzs2_trans;

    hipMalloc((void**)&cuda_Rows, sizeof(int)*(rows_init + 1));
    hipMalloc((void**)&cuda_Cols, sizeof(int)*nnzs_init);
    hipMalloc((void**)&cuda_Vals, sizeof(float)*nnzs_init);
    hipMalloc((void**)&cuda_Rows_Trans, sizeof(int)*(rows_init_trans + 1));
    hipMalloc((void**)&cuda_Cols_Trans, sizeof(int)*nnzs_init_trans);
    hipMalloc((void**)&cuda_Vals_Trans, sizeof(float)*nnzs_init_trans);
    hipMalloc((void**)&cuda_f, sizeof(float)*cols);
    hipMalloc((void**)&cuda_g, sizeof(int)*rows);
    hipMalloc((void**)&cuda_norm, sizeof(float)*cols);
    hipMalloc((void**)&cuda_bwproj, sizeof(float)*cols);
    hipMalloc((void**)&cuda_temp, sizeof(float)*rows);
    // printf("    End  : Allocate GPU Storage\n");

    // value initialization
    // printf("    Begin: GPU Storage Initialization\n");
    hipMemcpy(cuda_g, g, sizeof(int)* rows, hipMemcpyHostToDevice);
    hipMemcpy(cuda_norm, norm, sizeof(float)* cols, hipMemcpyHostToDevice);
    hipMemset(cuda_bwproj, 0, sizeof(float)*cols);
    hipMemset(cuda_temp, 0, sizeof(float)*rows);
    // hipMemset(cuda_f, init, sizeof(float)*cols);
    // hipMemsetD32(cuda_f, __float_as_int(init), cols);
    hipMemcpy(cuda_f, f, sizeof(float)* cols, hipMemcpyHostToDevice);
    hipMemset(cuda_Rows_Trans, 0, sizeof(int)*(cols+1));
    hipMemset(cuda_Cols_Trans, 0, sizeof(int)*nnzs);
    hipMemset(cuda_Vals_Trans, 0, sizeof(float)*nnzs);
    // printf("    End  : GPU Storage Initialization\n");

    
    // Determine grid size and section size (block size is set to 1024 by default)
    int blocksize = 1024;
    int gridsize_correl = ceil((double)rows / blocksize);
    int gridsize_update = ceil((double)cols / blocksize);
    int items_fwproj1 = halfrows1 + halfnnzs1;
    int items_fwproj2 = halfrows2 + halfnnzs1;
    int items_bwproj1 = halfrows1_trans + halfnnzs1_trans;
    int items_bwproj2 = halfrows2_trans + halfnnzs2_trans;
    int gridsize_fwproj1 = ceil(sqrt((double)items_fwproj1 / blocksize));
    int gridsize_fwproj2 = ceil(sqrt((double)items_fwproj2 / blocksize));
    int gridsize_bwproj1 = ceil(sqrt((double)items_bwproj1 / blocksize));
    int gridsize_bwproj2 = ceil(sqrt((double)items_bwproj2 / blocksize));
    int secsize_fwproj1 = ceil((double)items_fwproj1 / (blocksize * gridsize_fwproj1));
    int secsize_fwproj2 = ceil((double)items_fwproj2 / (blocksize * gridsize_fwproj2));
    int secsize_bwproj1 = ceil((double)items_bwproj1 / (blocksize * gridsize_bwproj1));
    int secsize_bwproj2 = ceil((double)items_bwproj2 / (blocksize * gridsize_bwproj2));

    clock_t initEnd = clock();
    printf("    End  : Initialization\n");
    double initTime = ((double) (initEnd - initStart)) / CLOCKS_PER_SEC;
    printf("    Elapsed time for initialization: %f\n\n", initTime);
    
    // iterations
    printf("    Begin: Iterations %d\n", Iterations);
    clock_t iterStart = clock();
    for(int i = 0; i < Iterations; i++){
        // forward projection for first half matrix
        csr_Rows[halfrows1] = offset;
        hipMemcpy(cuda_Rows, csr_Rows, sizeof(int)*(halfrows1 + 1), hipMemcpyHostToDevice);
        hipMemcpy(cuda_Cols, csr_Cols, sizeof(int)* halfnnzs1, hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals, csr_Vals, sizeof(float)* halfnnzs1, hipMemcpyHostToDevice);
        calcFwProj <<< gridsize_fwproj1, blocksize >>> (cuda_Rows, cuda_Vals, cuda_Cols, cuda_f, cuda_temp, secsize_fwproj1, halfrows1, halfnnzs1);

        // forward projection for second half matrix
        csr_Rows[halfrows1] = 0;
        hipMemcpy(cuda_Rows, csr_Rows+halfrows1, sizeof(int)*(halfrows2 + 1), hipMemcpyHostToDevice);
        hipMemcpy(cuda_Cols, csr_Cols+halfnnzs1, sizeof(int)* halfnnzs2, hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals, csr_Vals+halfnnzs1, sizeof(float)* halfnnzs2, hipMemcpyHostToDevice);
        calcFwProj <<< gridsize_fwproj2, blocksize >>> (cuda_Rows, cuda_Vals, cuda_Cols, cuda_f, cuda_temp+halfrows1, secsize_fwproj2, halfrows2, halfnnzs2);
        
        // correlation
        calcCorrel <<< gridsize_correl, blocksize >>> (cuda_g, cuda_temp, rows);

        // backward projection for first half transposed matrix
        csr_Rows_Trans[halfrows1_trans] = offset_trans;
        hipMemcpy(cuda_Rows_Trans, csr_Rows_Trans, sizeof(int)*(halfrows1_trans + 1), hipMemcpyHostToDevice);
        hipMemcpy(cuda_Cols_Trans, csr_Cols_Trans, sizeof(int)* halfnnzs1_trans, hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals_Trans, csr_Vals_Trans, sizeof(float)* halfnnzs1_trans, hipMemcpyHostToDevice);
        calcBwProj <<< gridsize_bwproj1, blocksize >>> (cuda_Rows_Trans, cuda_Vals_Trans, cuda_Cols_Trans, cuda_temp, cuda_bwproj, secsize_bwproj1, halfrows1_trans, halfnnzs1_trans);

        // backward projection for second half transposed matrix
        csr_Rows_Trans[halfrows1_trans] = 0;
        hipMemcpy(cuda_Rows_Trans, csr_Rows_Trans+halfrows1_trans, sizeof(int)*(halfrows2_trans + 1), hipMemcpyHostToDevice);
        hipMemcpy(cuda_Cols_Trans, csr_Cols_Trans+halfnnzs1_trans, sizeof(int)* halfnnzs2_trans, hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals_Trans, csr_Vals_Trans+halfnnzs1_trans, sizeof(float)* halfnnzs2_trans, hipMemcpyHostToDevice);
        calcBwProj <<< gridsize_bwproj2, blocksize >>> (cuda_Rows_Trans, cuda_Vals_Trans, cuda_Cols_Trans, cuda_temp, cuda_bwproj+halfrows1_trans, secsize_bwproj2, halfrows2_trans, halfnnzs2_trans);
        
        // update, for mlem naive calcUpdateAndClearBwproj should be used
        calcUpdateInPlace <<< gridsize_update, blocksize >>> (cuda_f, cuda_norm, cuda_bwproj, cols);
        
        // clear cuda_temp and cuda_bwproj
        hipMemset(cuda_temp,   0, sizeof(float)*rows);
        hipMemset(cuda_bwproj, 0, sizeof(float)*cols); 
    }
    clock_t iterEnd = clock();
    printf("    End  : Iterations %d\n", Iterations);
    double itertime = ((double) (iterEnd - iterStart)) / CLOCKS_PER_SEC;
    printf("    Elapsed time for iterations: %f\n", itertime);

    // Result is copied to f
    hipMemcpy(f, cuda_f, sizeof(float)*cols, hipMemcpyDeviceToHost);

    // free all memory
    if(cuda_Rows) hipFree(cuda_Rows);
    if(cuda_Cols) hipFree(cuda_Cols);
    if(cuda_Vals) hipFree(cuda_Vals);
    if(cuda_Rows_Trans) hipFree(cuda_Rows_Trans);
    if(cuda_Cols_Trans) hipFree(cuda_Cols_Trans);
    if(cuda_Vals_Trans) hipFree(cuda_Vals_Trans);
    if(cuda_g) hipFree(cuda_g);
    if(cuda_norm) hipFree(cuda_norm);
    if(cuda_f) hipFree(cuda_f);
    if(cuda_bwproj) hipFree(cuda_bwproj);
    if(cuda_temp) hipFree(cuda_temp);
    

    clock_t end = clock();
    double totaltime = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("    Elapsed time totally       : %f\n\n", totaltime);
}


int main(){
    // host variables
    int *csr_Rows, *csr_Cols, *csr_Rows_Trans, *csr_Cols_Trans, *g, rows, cols, nnzs, sum_g = 0;
    float *csr_Vals, *csr_Vals_Trans, *f, *norm, sum_norm = 0.0f;


    // read matrix
    printf("\nBegin: Read Matrix\n");
    Csr4Matrix matrix("/scratch/pet/madpet2.p016.csr4.small");
    printf("End  : Read Matrix\n\n");
    printf("Begin: Create CSR Format for Matrix\n");
    clock_t start = clock();
    rows = matrix.rows();
    cols = matrix.columns();
    nnzs = matrix.elements();
    printf("    The matrix contains %d rows, %d cols, %d nnzs\n", rows, cols, nnzs);
    matrix.mapRows(0, rows);    
    csr_Rows = (int*)malloc(sizeof(int) * (rows + 1));
    csr_Cols = (int*)malloc(sizeof(int) * nnzs);
    csr_Vals = (float*)malloc(sizeof(float) * nnzs);
    csr_format_for_cuda(matrix, csr_Vals, csr_Rows, csr_Cols);
    Vector<float> norm_helper(cols, 0.0);
    calcColumnSums(matrix, norm_helper);
    norm = norm_helper.ptr();
    // TODO: calculate sum_norm using gpu
    for(int i = 0; i < cols; i++)
        sum_norm += norm[i];
    clock_t end = clock();
    printf("End  : Create CSR Format for Matrix\n");
    double elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time for creating CSR: %f\n\n", elapsed);
    

    // read image
    printf("Begin: Read Image\n");
    start = clock();
    Vector<int> image("/scratch/pet/Trues_Derenzo_GATE_rot_sm_200k.LMsino.small");
    g = image.ptr();
    // TODO: calculate sum_g using gpu
    for(int i = 0; i < rows; i++)
        sum_g += g[i];
    end = clock();
    printf("End  : Read Image\n");
    elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time for reading image: %f\n\n", elapsed);

    // calculate initial value
    float init = sum_g / sum_norm;
    printf("Sum of norms: %f\n", sum_norm);
    printf("Sum of g    : %d\n", sum_g);
    printf("Initial f   : %f\n\n", init);
    f = (float*)malloc(sizeof(float)*cols);
    for(int i = 0; i < cols; i++)
        f[i] = init;
    

    // transpose matrix
    printf("Begin: Transpose Matrix\n");
    start = clock();
    // transpose matrix using GPU
    // transposeCSR(cuda_Rows, cuda_Cols, cuda_Vals, cuda_Rows_Trans, cuda_Cols_Trans, cuda_Vals_Trans, rows, cols, nnzs);
    
    // transpose matrix using CPU
    csr_Rows_Trans = (int*) calloc (cols+1,sizeof(int));
    csr_Cols_Trans = (int*) calloc (nnzs,sizeof(int));
    csr_Vals_Trans = (float*) calloc (nnzs,sizeof(float));
    sptrans_scanTrans<int, float>(rows, cols, nnzs, csr_Rows, csr_Cols, csr_Vals, csr_Cols_Trans, csr_Rows_Trans, csr_Vals_Trans);
    end = clock();
    printf("End  : Transpose Matrix\n");
    elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time for transposing matrix: %f\n\n", elapsed);

    
    // run mlem algorithm matrix
    printf("\n***********************************************\n");
    printf("Begin: Run MLEM for %d iterations\n", Iterations);
    if(MLEM_Version == 0)
        mlem_naive(csr_Rows, csr_Cols, csr_Vals, csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, g, norm, f, rows, cols, nnzs);
    else 
        mlem_nccl(csr_Rows, csr_Cols, csr_Vals, csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, g, norm, f, rows, cols, nnzs);
    printf("End  : Run MLEM for %d iterations\n", Iterations);
    printf("***********************************************\n");

    // sum up all elements in the solution f
    float sum = 0;
    for(int i = 0; i < cols; i++)
        sum += f[i];
    
    printf("\nSum f: %f\n\n", sum);
    
    if (csr_Rows) free(csr_Rows);
    if (csr_Cols) free(csr_Cols);
    if (csr_Vals) free(csr_Vals);
    if (csr_Rows_Trans) free(csr_Rows_Trans);
    if (csr_Cols_Trans) free(csr_Cols_Trans);
    if (csr_Vals_Trans) free(csr_Vals_Trans);
    // if (g) free(g);
    // if (norm) free(norm);
    if (f) free(f);

    return 0;
}
