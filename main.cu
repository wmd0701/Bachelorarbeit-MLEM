#include "hip/hip_runtime.h"
#include "algorithm"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "math.h"
#include "kernel.cuh"
#include "hipsparse.h"
#include "csr4matrix.hpp"
#include "vector.hpp"
#include "time.h"
#include "sptrans.h"

// #define TransposeMatrixUsingCPU true
#define Iterations 300

void csr_format_for_cuda(const Csr4Matrix& matrix, float* csrVal, int* csrRowInd, int* csrColInd){   
    int index = 0;
    csrRowInd[index] = 0;
// pragma omp parallel for schedule (static)
    for (int row = 0; row < matrix.rows(); ++row) {
        csrRowInd[row + 1] = csrRowInd[row] + (int)matrix.elementsInRow(row);
	
        std::for_each(matrix.beginRow2(row), matrix.endRow2(row),[&](const RowElement<float>& e){ 
            csrVal[index] = e.value();
            csrColInd[index] = (int)e.column() ;
            index = index + 1; }
        );
    }
}

void calcColumnSums(const Csr4Matrix& matrix, Vector<float>& norm)
{
    assert(matrix.columns() == norm.size());

    std::fill(norm.ptr(), norm.ptr() + norm.size(), 0.0);
    matrix.mapRows(0, matrix.rows());

  // pragma omp parallel for schedule (static)
    for (uint32_t row=0; row<matrix.rows(); ++row) {
        std::for_each(matrix.beginRow2(row), matrix.endRow2(row),
                      [&](const RowElement<float>& e){ norm[e.column()] += e.value(); });
    }
    // norm.writeToFile("norm-0.out");
}

void transposeCSR(int *cuda_Rows, int *cuda_Cols, float *cuda_Vals, int *cuda_Rows_Trans, int *cuda_Cols_Trans, float *cuda_Vals_Trans,
                    int rows, int cols, int nnzs){
    hipsparseStatus_t status;
	hipsparseHandle_t handle = 0;
	status = hipsparseCreate(&handle);
	if (status != HIPSPARSE_STATUS_SUCCESS){
        hipError_t cuda_err = hipGetLastError();
        printf("    Fail : CSR to CSC, cusparese initialization failed , ERROR %d, %s\n", status, hipGetErrorString(cuda_err));
    }
    status = hipsparseScsr2csc(handle, rows, cols, nnzs, cuda_Vals, cuda_Rows, cuda_Cols, cuda_Vals_Trans, cuda_Cols_Trans, cuda_Rows_Trans, 
                                HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
    if (status != HIPSPARSE_STATUS_SUCCESS)
        printf("    Fail : CSR to CSC, cusparse transpose failed\n");

    status = hipsparseDestroy(handle);
    handle = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS)
        printf("    Fail : CSR to CSC, cusparse destroy failed\n");
    
    // cusparse functions are asynchronous
    hipDeviceSynchronize();
}


// return row index for in which row the nnzs are distributed into two pars equally
int halfMatrix(int *csr_Rows, int nnzs, int rows){
    int i = 0;
    int halfnnzs = nnzs / 2;
    for(; i <= rows; i++)
        if(csr_Rows[i] > halfnnzs)
            break;
    return i;
}

void mlem(  int *csr_Rows, float *csr_Vals, int *csr_Cols, 
            int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, 
            int *g, float *norm, float *f, int rows, int cols, int nnzs){
    clock_t start = clock();
    
    // halve the matrix
    int halfrows1 = halfMatrix(csr_Rows, nnzs, rows);
    int halfrows2 = rows - halfrows1;
    int halfnnzs1 = csr_Rows[halfrows1];
    int halfnnzs2 = nnzs - halfnnzs1;
    int offset = csr_Rows[halfrows1];
    printf("First  half matrix contains %d rows and %d nnzs\n", halfrows1, halfnnzs1);
    printf("Second half matrix contains %d rows and %d nnzs\n", halfrows2, halfnnzs2);
    // adjust row array for the second half matrix
    // TODO: accelerate this adjustment with GPU
    for(int i = halfrows1+1; i <= rows; i++)
        csr_Rows[i] -= offset;

    // halve the transposed matrix
    int halfrows1_trans = halfMatrix(csr_Rows_Trans, nnzs, cols);
    int halfrows2_trans = cols - halfrows1_trans;
    int halfnnzs1_trans = csr_Rows_Trans[halfrows1_trans];
    int halfnnzs2_trans = nnzs - halfnnzs1_trans;
    int offset_trans = csr_Rows_Trans[halfrows1_trans];
    printf("First  half transposed matrix contains %d rows and %d nnzs\n", halfrows1_trans, halfnnzs1_trans);
    printf("Second half transposed matrix contains %d rows and %d nnzs\n", halfrows2_trans, halfnnzs2_trans);
    // adjust row array for the second half matrix
    // TODO: accelerate this adjustment with GPU
    for(int i = halfrows1_trans+1; i <= cols; i++)
        csr_Rows_Trans[i] -= offset_trans;


    // device variables
    int *cuda_Rows, *cuda_Cols, *cuda_Rows_Trans, *cuda_Cols_Trans, *cuda_g;
    float *cuda_Vals, *cuda_Vals_Trans, *cuda_norm, *cuda_bwproj, *cuda_temp, *cuda_f;


    // allocate device storage
    printf("    Begin: Allocate GPU Storage\n");
    int rows_init = halfrows1 > halfrows2 ? halfrows1 : halfrows2;
    int nnzs_init = halfnnzs1 > halfnnzs2 ? halfnnzs1 : halfnnzs2;
    int rows_init_trans = halfrows1_trans > halfrows2_trans ? halfrows1_trans : halfrows2_trans;
    int nnzs_init_trans = halfnnzs1_trans > halfnnzs2_trans ? halfnnzs1_trans : halfnnzs2_trans;

    hipMalloc((void**)&cuda_Rows, sizeof(int)*(rows_init + 1));
    hipMalloc((void**)&cuda_Cols, sizeof(int)*nnzs_init);
    hipMalloc((void**)&cuda_Vals, sizeof(float)*nnzs_init);
    hipMalloc((void**)&cuda_Rows_Trans, sizeof(int)*(rows_init_trans + 1));
    hipMalloc((void**)&cuda_Cols_Trans, sizeof(int)*nnzs_init_trans);
    hipMalloc((void**)&cuda_Vals_Trans, sizeof(float)*nnzs_init_trans);
    hipMalloc((void**)&cuda_f, sizeof(float)*cols);
    hipMalloc((void**)&cuda_g, sizeof(int)*rows);
    hipMalloc((void**)&cuda_norm, sizeof(float)*cols);
    hipMalloc((void**)&cuda_bwproj, sizeof(float)*cols);
    hipMalloc((void**)&cuda_temp, sizeof(float)*rows);
    printf("    End  : Allocate GPU Storage\n");

    // value initialization
    printf("    Begin: GPU Storage Initialization\n");
    hipMemcpy(cuda_g, g, sizeof(int)* rows, hipMemcpyHostToDevice);
    hipMemcpy(cuda_norm, norm, sizeof(float)* cols, hipMemcpyHostToDevice);
    hipMemset(cuda_bwproj, 0, sizeof(float)*cols);
    hipMemset(cuda_temp, 0, sizeof(float)*rows);
    // hipMemset(cuda_f, init, sizeof(float)*cols);
    // hipMemsetD32(cuda_f, __float_as_int(init), cols);
    hipMemcpy(cuda_f, f, sizeof(float)* cols, hipMemcpyHostToDevice);
    hipMemset(cuda_Rows_Trans, 0, sizeof(int)*(cols+1));
    hipMemset(cuda_Cols_Trans, 0, sizeof(int)*nnzs);
    hipMemset(cuda_Vals_Trans, 0, sizeof(float)*nnzs);
    printf("    End  : GPU Storage Initialization\n");

    
    // Determine grid size and section size (block size is set to 1024 by default)
    int blocksize = 1024;
    int gridsize_correl = ceil((double)rows / blocksize);
    int gridsize_update = ceil((double)cols / blocksize);
    int items_fwproj1 = halfrows1 + halfnnzs1;
    int items_fwproj2 = halfrows2 + halfnnzs1;
    int items_bwproj1 = halfrows1_trans + halfnnzs1_trans;
    int items_bwproj2 = halfrows2_trans + halfnnzs2_trans;
    int gridsize_fwproj1 = ceil(sqrt((double)items_fwproj1 / blocksize));
    int gridsize_fwproj2 = ceil(sqrt((double)items_fwproj2 / blocksize));
    int gridsize_bwproj1 = ceil(sqrt((double)items_bwproj1 / blocksize));
    int gridsize_bwproj2 = ceil(sqrt((double)items_bwproj2 / blocksize));
    int secsize_fwproj1 = ceil((double)items_fwproj1 / (blocksize * gridsize_fwproj1));
    int secsize_fwproj2 = ceil((double)items_fwproj2 / (blocksize * gridsize_fwproj2));
    int secsize_bwproj1 = ceil((double)items_bwproj1 / (blocksize * gridsize_bwproj1));
    int secsize_bwproj2 = ceil((double)items_bwproj2 / (blocksize * gridsize_bwproj2));

    
    // iterations
    printf("    Begin: Iterations %d\n", Iterations);
    clock_t startIter = clock();
    for(int i = 0; i < Iterations; i++){
        // forward projection for first half matrix
        csr_Rows[halfrows1] = offset;
        hipMemcpy(cuda_Rows, csr_Rows, sizeof(int)*(halfrows1 + 1), hipMemcpyHostToDevice);
        hipMemcpy(cuda_Cols, csr_Cols, sizeof(int)* halfnnzs1, hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals, csr_Vals, sizeof(float)* halfnnzs1, hipMemcpyHostToDevice);
        calcFwProj <<< gridsize_fwproj1, blocksize >>> (cuda_Rows, cuda_Vals, cuda_Cols, cuda_f, cuda_temp, secsize_fwproj1, halfrows1, halfnnzs1);

        // forward projection for second half matrix
        csr_Rows[halfrows1] = 0;
        hipMemcpy(cuda_Rows, csr_Rows+halfrows1, sizeof(int)*(halfrows2 + 1), hipMemcpyHostToDevice);
        hipMemcpy(cuda_Cols, csr_Cols+halfnnzs1, sizeof(int)* halfnnzs2, hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals, csr_Vals+halfnnzs1, sizeof(float)* halfnnzs2, hipMemcpyHostToDevice);
        calcFwProj <<< gridsize_fwproj2, blocksize >>> (cuda_Rows, cuda_Vals, cuda_Cols, cuda_f, cuda_temp+halfrows1, secsize_fwproj2, halfrows2, halfnnzs2);
        
        // correlation
        calcCorrel <<< gridsize_correl, blocksize >>> (cuda_g, cuda_temp, rows);

        // backward projection for first half transposed matrix
        csr_Rows_Trans[halfrows1_trans] = offset_trans;
        hipMemcpy(cuda_Rows_Trans, csr_Rows_Trans, sizeof(int)*(halfrows1_trans + 1), hipMemcpyHostToDevice);
        hipMemcpy(cuda_Cols_Trans, csr_Cols_Trans, sizeof(int)* halfnnzs1_trans, hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals_Trans, csr_Vals_Trans, sizeof(float)* halfnnzs1_trans, hipMemcpyHostToDevice);
        calcBkProj <<< gridsize_bwproj1, blocksize >>> (cuda_Rows_Trans, cuda_Vals_Trans, cuda_Cols_Trans, cuda_temp, cuda_bwproj, secsize_bwproj1, halfrows1_trans, halfnnzs1_trans);

        // backward projection for second half transposed matrix
        csr_Rows_Trans[halfrows1_trans] = 0;
        hipMemcpy(cuda_Rows_Trans, csr_Rows_Trans+halfrows1_trans, sizeof(int)*(halfrows2_trans + 1), hipMemcpyHostToDevice);
        hipMemcpy(cuda_Cols_Trans, csr_Cols_Trans+halfnnzs1_trans, sizeof(int)* halfnnzs2_trans, hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals_Trans, csr_Vals_Trans+halfnnzs1_trans, sizeof(float)* halfnnzs2_trans, hipMemcpyHostToDevice);
        calcBkProj <<< gridsize_bwproj2, blocksize >>> (cuda_Rows_Trans, cuda_Vals_Trans, cuda_Cols_Trans, cuda_temp, cuda_bwproj+halfrows1_trans, secsize_bwproj2, halfrows2_trans, halfnnzs2_trans);
        
        // update
        calcUpdate <<< gridsize_update, blocksize >>> (cuda_f, cuda_norm, cuda_bwproj, cols);
        
        // clear temp vector
        clearTemp  <<< gridsize_correl, blocksize >>> (cuda_temp, rows);
    }
    clock_t endIter = clock();
    printf("    End  : Iterations %d\n\n", Iterations);
    
    // Result is copied to f
    hipMemcpy(f, cuda_f, sizeof(float)*cols, hipMemcpyDeviceToHost);

    // free all memory
    if(cuda_Rows) hipFree(cuda_Rows);
    if(cuda_Cols) hipFree(cuda_Cols);
    if(cuda_Vals) hipFree(cuda_Vals);
    if(cuda_Rows_Trans) hipFree(cuda_Rows_Trans);
    if(cuda_Cols_Trans) hipFree(cuda_Cols_Trans);
    if(cuda_Vals_Trans) hipFree(cuda_Vals_Trans);
    if(cuda_g) hipFree(cuda_g);
    if(cuda_norm) hipFree(cuda_norm);
    if(cuda_f) hipFree(cuda_f);
    if(cuda_bwproj) hipFree(cuda_bwproj);
    if(cuda_temp) hipFree(cuda_temp);
    

    clock_t end = clock();
    double totaltime = ((double) (end - start)) / CLOCKS_PER_SEC;
    double itertime = ((double) (endIter - startIter)) / CLOCKS_PER_SEC;
    printf("    Time for the whole MLEM function: %f\n", totaltime);
    printf("    Time for the MLEM iterations: %f\n\n", itertime);
}


int main(){

    // host variables
    int *csr_Rows, *csr_Cols, *g, rows, cols, nnzs, sum_g = 0;
    float *csr_Vals, *norm, sum_norm = 0.0f;


    // read matrix
    printf("Begin: Read Matrix\n");
    Csr4Matrix matrix("/scratch/pet/madpet2.p016.csr4.small");
    printf("End  : Read Matrix\n");
    printf("Begin: Create CSR Format for Matrix\n");
    rows = matrix.rows();
    cols = matrix.columns();
    nnzs = matrix.elements();
    matrix.mapRows(0, rows);    
    csr_Rows = (int*)malloc(sizeof(int) * (rows + 1));
    csr_Cols = (int*)malloc(sizeof(int) * nnzs);
    csr_Vals = (float*)malloc(sizeof(float) * nnzs);
    csr_format_for_cuda(matrix, csr_Vals, csr_Rows, csr_Cols);
    Vector<float> norm_helper(cols, 0.0);
    calcColumnSums(matrix, norm_helper);
    norm = norm_helper.ptr();
    // TODO: calculate sum_norm using gpu
    for(int i = 0; i < cols; i++)
        sum_norm += norm[i];
    printf("End  : Create CSR Format for Matrix\n");
    

    // read image
    printf("Begin: Read Image\n");
    Vector<int> image("/scratch/pet/Trues_Derenzo_GATE_rot_sm_200k.LMsino.small");
    g = image.ptr();
    // TODO: calculate sum_g using gpu
    for(int i = 0; i < rows; i++)
        sum_g += g[i];
    printf("End  : Read Image\n\n");
    

    float init = sum_g / sum_norm;
    printf("Sum of norms: %f\n", sum_norm);
    printf("Sum of g    : %d\n", sum_g);
    printf("Initial f   : %f\n\n", init);


    // transpose matrix
    printf("Begin: Transpose Matrix\n");
    // transpose matrix using GPU
    // transposeCSR(cuda_Rows, cuda_Cols, cuda_Vals, cuda_Rows_Trans, cuda_Cols_Trans, cuda_Vals_Trans, rows, cols, nnzs);
    
    // transpose matrix using CPU
    int *csr_Rows_Trans = (int*) calloc (cols+1,sizeof(int));
    int *csr_Cols_Trans = (int*) calloc (nnzs,sizeof(int));
    float *csr_Vals_Trans = (float*) calloc (nnzs,sizeof(float));
    sptrans_scanTrans<int, float>(rows, cols, nnzs, csr_Rows, csr_Cols, csr_Vals, csr_Cols_Trans, csr_Rows_Trans, csr_Vals_Trans);
    printf("End  : Transpose Matrix\n");

    float *f = (float*)malloc(sizeof(float)*cols);
    for(int i = 0; i < cols; i++)
        f[i] = init;
    
    // run mlem algorithm matrix
    printf("\n\n******************************\n");
    printf("Begin: Run MLEM for %d iterations\n", Iterations);
    mlem(csr_Rows, csr_Vals, csr_Cols, csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, g, norm, f, rows, cols, nnzs);
    printf("End  : Run MLEM for %d iterations\n", Iterations);
    printf("******************************\n");

    // sum up all elements in the solution f
    float sum = 0;
    for(int i = 0; i < cols; i++)
        sum += f[i];
    
    printf("\nSum f: %f\n\n", sum);
    
    if (csr_Rows) free(csr_Rows);
    if (csr_Cols) free(csr_Cols);
    if (csr_Vals) free(csr_Vals);
    if (csr_Rows_Trans) free(csr_Rows_Trans);
    if (csr_Cols_Trans) free(csr_Cols_Trans);
    if (csr_Vals_Trans) free(csr_Vals_Trans);
    // if (g) free(g);
    // if (norm) free(norm);
    if (f) free(f);

    return 0;
}
