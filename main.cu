#include "hip/hip_runtime.h"
#include "algorithm"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "math.h"
#include "kernel.cuh"
#include "hipsparse.h"
#include "csr4matrix.hpp"
#include "vector.hpp"
#include "time.h"
#include "sptrans.h"
#include "nccl.h"

void csr_format_for_cuda(const Csr4Matrix& matrix, float* csrVal, unsigned long* csrRowInd, unsigned int* csrColInd);
void calcColumnSums(const Csr4Matrix& matrix, Vector<float>& norm);
void partitionMatrix(unsigned long *csr_Rows, unsigned long nnzs, unsigned int rows, unsigned int device_numbers, unsigned int *segments, unsigned int *segment_rows, unsigned int *segment_nnzs, unsigned long *offsets);
void mlem(unsigned long *csr_Rows, unsigned int *csr_Cols, float *csr_Vals, unsigned long *csr_Rows_Trans, unsigned int *csr_Cols_Trans, float *csr_Vals_Trans, int *g, float *norm, float *f, float *result_f, unsigned int rows, unsigned int cols, unsigned long nnzs, unsigned int iterations, unsigned int device_numbers, unsigned int matrix_vector_mul, unsigned int secsize_fp, unsigned int secsize_bp, unsigned int using_trans);


/*
    argv[1]: path for matrix
    argv[2]: path for image
    argv[3]: iteration times
    argv[4]: number of GPUs to be used
    argv[5]: section size for forward projection in NVIDIA merge-based
    argv[6]: section size for backward projection in NVIDIA merge-based
    argv[7]: whether to use transposed matrix              0: use transposed matrix    1: not use transposed matrix
    argv[8]: which matrix-vector multiplication to use     0: NVIDIA merge-based       1: coalesced brutal warp

    run examples:
    ./test /scratch/pet/madpet2.p016.csr4.small /scratch/pet/Trues_Derenzo_GATE_rot_sm_200k.LMsino.small 500 2 4 4 0 0
    ./test /scratch/pet/madpet2.p016.csr4.small /scratch/pet/Trues_Derenzo_GATE_rot_sm_200k.LMsino.small 500 2 3 9 1 0
    ./test /scratch/pet/madpet2.p016.csr4.small /scratch/pet/Trues_Derenzo_GATE_rot_sm_200k.LMsino.small 500 2 3 9 0 1
    ./test /scratch/pet/madpet2.p016.csr4.small /scratch/pet/Trues_Derenzo_GATE_rot_sm_200k.LMsino.small 500 2 5 5 1 1
    ./test /scratch/pet/madpet2.p016.csr4 /scratch/pet/Trues_Derenzo_GATE_rot_sm_200k.LMsino 500 1 5 5 1 1
*/
int main(int argc, char **argv){
    if(argc != 9){
        printf("Too less or too many parameters for main function! Program exits!\n");
        return 0;
    }

    std::string matrixPath(argv[1]);
    std::string imagePath(argv[2]);
    unsigned int iterations          = strtol(argv[3], NULL, 10);
    unsigned int device_numbers      = strtol(argv[4], NULL, 10);
    unsigned int secsize_fp          = strtol(argv[5], NULL, 10);
    unsigned int secsize_bp          = strtol(argv[6], NULL, 10);
    unsigned int using_trans         = strtol(argv[7], NULL, 10);
    unsigned int matrix_vector_mul   = strtol(argv[8], NULL, 10);
    
    int device_numbers_available = 0;
    hipGetDeviceCount(&device_numbers_available);
    if(device_numbers_available < device_numbers){
        printf("Number of available GPUs less than ordered! Program exits!\n");
        return 0;
    }

    // host variables
    unsigned long *csr_Rows, *csr_Rows_Trans, nnzs;
    unsigned int  *csr_Cols, *csr_Cols_Trans, rows, cols;
    int *g, sum_g = 0;
    float *csr_Vals, *csr_Vals_Trans, *f, *result_f, *norm, sum_norm = 0.0f;

    // read matrix
    Csr4Matrix matrix(matrixPath);
    rows = matrix.rows();
    cols = matrix.columns();
    nnzs = matrix.elements();
    matrix.mapRows(0, rows);    
    csr_Rows = (unsigned long*)malloc(sizeof(unsigned long) * (rows + 1));
    csr_Cols = (unsigned int*)malloc(sizeof(unsigned int) * nnzs);
    csr_Vals = (float*)malloc(sizeof(float) * nnzs);
    csr_format_for_cuda(matrix, csr_Vals, csr_Rows, csr_Cols);
    Vector<float> norm_helper(cols, 0.0);
    calcColumnSums(matrix, norm_helper);
    norm = norm_helper.ptr();
    for(unsigned int i = 0; i < cols; i++)
        sum_norm += norm[i];
    
    // read image
    Vector<int> image(imagePath);
    g = image.ptr();
    for(unsigned int i = 0; i < rows; i++)
        sum_g += g[i];
    
    // calculate initial value
    float init = sum_g / sum_norm;
    f = (float*)malloc(sizeof(float)*cols);
    result_f = (float*)malloc(sizeof(float)*cols);
    for(unsigned int i = 0; i < cols; i++)
        f[i] = init;

    // transpose matrix using algorithm ScanTrans, working on CPU
    if(using_trans == 0){
        csr_Rows_Trans = (unsigned long*) calloc (cols+1,sizeof(unsigned long));
        csr_Cols_Trans = (unsigned int*) calloc (nnzs,sizeof(unsigned int));
        csr_Vals_Trans = (float*) calloc (nnzs,sizeof(float));
        sptrans_scanTrans_specialized(rows, cols, nnzs, csr_Rows, csr_Cols, csr_Vals, csr_Cols_Trans, csr_Rows_Trans, csr_Vals_Trans);
    }

        
    // run MLEM
    mlem(   csr_Rows, 
            csr_Cols,
            csr_Vals,
            csr_Rows_Trans,
            csr_Cols_Trans,
            csr_Vals_Trans,
            g,
            norm,
            f, 
            result_f,
            rows,
            cols,
            nnzs,
            iterations,
            device_numbers,
            matrix_vector_mul,
            secsize_fp,
            secsize_bp,
            using_trans );
    
    // clear storage
    if (csr_Rows) free(csr_Rows);
    if (csr_Cols) free(csr_Cols);
    if (csr_Vals) free(csr_Vals);
    // if (g) free(g);
    // if (norm) free(norm);
    if (f) free(f);
    if(result_f) free(result_f);
    if(using_trans == 0){
        if (csr_Rows_Trans) free(csr_Rows_Trans);
        if (csr_Cols_Trans) free(csr_Cols_Trans);
        if (csr_Vals_Trans) free(csr_Vals_Trans);
    }

    return 0;
}

void mlem(  unsigned long *csr_Rows, 
            unsigned int *csr_Cols, 
            float *csr_Vals, 
            unsigned long *csr_Rows_Trans, 
            unsigned int *csr_Cols_Trans, 
            float *csr_Vals_Trans, 
            int *g, 
            float *norm, 
            float *f, 
            float *result_f, 
            unsigned int rows, 
            unsigned int cols, 
            unsigned long nnzs, 
            unsigned int iterations, 
            unsigned int device_numbers, 
            unsigned int matrix_vector_mul, 
            unsigned int secsize_fp,
            unsigned int secsize_bp,
            unsigned int using_trans )
{    
    // partition matrix
    unsigned int *segments = (unsigned int*)malloc((device_numbers+1)*sizeof(unsigned int));
    unsigned int *segment_rows = (unsigned int*)malloc(device_numbers*sizeof(unsigned int));
    unsigned int *segment_nnzs = (unsigned int*)malloc(device_numbers*sizeof(unsigned int));
    unsigned long *offsets = (unsigned long*)malloc(device_numbers*sizeof(unsigned long));
    partitionMatrix(csr_Rows, nnzs, rows, device_numbers, segments, segment_rows, segment_nnzs, offsets);

    // partition transposed matrix
    unsigned int *segments_trans;
    unsigned int *segment_rows_trans;
    unsigned int *segment_nnzs_trans;
    unsigned long *offsets_trans;
    if(using_trans == 0){
        segments_trans = (unsigned int*)malloc((device_numbers+1)*sizeof(unsigned int));
        segment_rows_trans = (unsigned int*)malloc(device_numbers*sizeof(unsigned int));
        segment_nnzs_trans = (unsigned int*)malloc(device_numbers*sizeof(unsigned int));
        offsets_trans = (unsigned long*)malloc(device_numbers*sizeof(unsigned long));
        partitionMatrix(csr_Rows_Trans, nnzs, cols, device_numbers, segments_trans, segment_rows_trans, segment_nnzs_trans, offsets_trans);
    }

    // NCCL components
    ncclComm_t *comms = (ncclComm_t*)malloc(device_numbers * sizeof(ncclComm_t));;
    hipStream_t *streams = (hipStream_t*)malloc(device_numbers * sizeof(hipStream_t));
    int *devices = (int*)malloc(device_numbers * sizeof(int));    

    // device variables
    unsigned int **cuda_Rows = (unsigned int**)malloc(device_numbers*sizeof(unsigned int*));
    unsigned int **cuda_Cols = (unsigned int**)malloc(device_numbers*sizeof(unsigned int*));
    int **cuda_g = (int**)malloc(device_numbers*sizeof(int*));
    float **cuda_Vals = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_norm = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_bwproj = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_temp = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_f = (float**)malloc(device_numbers*sizeof(float*));
    unsigned int **cuda_Rows_Trans;
    unsigned int **cuda_Cols_Trans;
    float **cuda_Vals_Trans;
    if(using_trans==0){
        cuda_Rows_Trans = (unsigned int**)malloc(device_numbers*sizeof(unsigned int*));
        cuda_Cols_Trans = (unsigned int**)malloc(device_numbers*sizeof(unsigned int*));
        cuda_Vals_Trans = (float**)malloc(device_numbers*sizeof(float*));
    }

    // initialization
    unsigned int blocksize = 1024;   // unique blocksize for all kernel calls
    unsigned int *gridsize_fwproj = (unsigned int*)malloc(device_numbers*sizeof(unsigned int));
    unsigned int *gridsize_correl = (unsigned int*)malloc(device_numbers*sizeof(unsigned int));
    unsigned int *gridsize_bwproj = (unsigned int*)malloc(device_numbers*sizeof(unsigned int));
    unsigned int *gridsize_update = (unsigned int*)malloc(device_numbers*sizeof(unsigned int));
    for(unsigned int i = 0; i < device_numbers; i++){
        hipSetDevice(i);
        hipStreamCreate(streams+i);
        devices[i] = i;

        hipMalloc((void**)&cuda_Rows[i], sizeof(unsigned int)*(segment_rows[i] + 1));
        hipMalloc((void**)&cuda_Cols[i], sizeof(unsigned int)*segment_nnzs[i]);
        hipMalloc((void**)&cuda_Vals[i], sizeof(float)*segment_nnzs[i]);
        if(using_trans == 0){
            hipMalloc((void**)&cuda_Rows_Trans[i], sizeof(unsigned int)*(segment_rows_trans[i] + 1));
            hipMalloc((void**)&cuda_Cols_Trans[i], sizeof(unsigned int)*segment_nnzs_trans[i]);
            hipMalloc((void**)&cuda_Vals_Trans[i], sizeof(float)*segment_nnzs_trans[i]);
        }
        hipMalloc((void**)&cuda_f[i], sizeof(float)*cols);
        hipMalloc((void**)&cuda_bwproj[i], sizeof(float)*cols);
        hipMalloc((void**)&cuda_g[i], sizeof(int)*segment_rows[i]);
        if(using_trans == 0){
            hipMalloc((void**)&cuda_temp[i], sizeof(float)*rows);
            hipMalloc((void**)&cuda_norm[i], sizeof(float)*segment_rows_trans[i]);
        }
        else{
            hipMalloc((void**)&cuda_temp[i], sizeof(float)*segment_rows[i]);
            hipMalloc((void**)&cuda_norm[i], sizeof(float)*cols);
        }
        
        
        // copy matrix from host to devices
        for(unsigned int j = segments[i]; j <= segments[i+1]; j++)
            csr_Rows[j] -= offsets[i];
        unsigned int *csr_Rows_help = (unsigned int*)malloc((segment_rows[i] + 1)*sizeof(unsigned int));
        for(unsigned int j = 0; j < segment_rows[i] + 1; j++)
            csr_Rows_help[j] = (unsigned int)csr_Rows[segments[i]+j];
        hipMemcpy(cuda_Rows[i], csr_Rows_help, sizeof(unsigned int)*(segment_rows[i] + 1), hipMemcpyHostToDevice);
        csr_Rows[segments[i+1]] += offsets[i];
        free(csr_Rows_help);
        hipMemcpy(cuda_Cols[i], csr_Cols+offsets[i], sizeof(unsigned int)*segment_nnzs[i], hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals[i], csr_Vals+offsets[i], sizeof(float)*segment_nnzs[i], hipMemcpyHostToDevice);

        // copy transposed matrix from host to devices
        if(using_trans == 0){
            for(unsigned int j = segments_trans[i]; j <= segments_trans[i+1]; j++)
                csr_Rows_Trans[j] -= offsets_trans[i];
            unsigned int *csr_Rows_Trans_help = (unsigned int*)malloc((segment_rows_trans[i] + 1)*sizeof(unsigned int));
            for(unsigned int j = 0; j < segment_rows_trans[i] + 1; j++)
                csr_Rows_Trans_help[j] = (unsigned int)csr_Rows_Trans[segments_trans[i]+j];
            hipMemcpy(cuda_Rows_Trans[i], csr_Rows_Trans_help, sizeof(unsigned int)*(segment_rows_trans[i] + 1), hipMemcpyHostToDevice);
            csr_Rows_Trans[segments_trans[i+1]] += offsets_trans[i];
            free(csr_Rows_Trans_help);
            hipMemcpy(cuda_Cols_Trans[i], csr_Cols_Trans+offsets_trans[i], sizeof(unsigned int)*segment_nnzs_trans[i], hipMemcpyHostToDevice);
            hipMemcpy(cuda_Vals_Trans[i], csr_Vals_Trans+offsets_trans[i], sizeof(float)*segment_nnzs_trans[i], hipMemcpyHostToDevice);
        }

        // copy other vectors from host to devices
        hipMemcpy(cuda_g[i], g+segments[i], sizeof(int)*segment_rows[i], hipMemcpyHostToDevice);
        hipMemcpy(cuda_f[i], f, sizeof(float)*cols, hipMemcpyHostToDevice);
        hipMemset(cuda_bwproj[i], 0, sizeof(float)*cols);
        if(using_trans == 0){
            hipMemset(cuda_temp[i], 0, sizeof(float)*rows);
            hipMemcpy(cuda_norm[i], norm+segments_trans[i], sizeof(float)*segment_rows_trans[i], hipMemcpyHostToDevice);
        }
        else{
            hipMemset(cuda_temp[i], 0, sizeof(float)*segment_rows[i]);
            hipMemcpy(cuda_norm[i], norm, sizeof(float)*cols, hipMemcpyHostToDevice);
        }
        

        // determine grid size for all MLEM steps
        gridsize_correl[i] = ceil((double)segment_rows[i] / blocksize);
        
        if(matrix_vector_mul == 0)
            gridsize_fwproj[i] = ceil(double(segment_rows[i] + segment_nnzs[i]) / (blocksize * secsize_fp));
        else
            gridsize_fwproj[i] = ceil(double(segment_rows[i]) / 32);
        
        if(using_trans == 0){
            gridsize_update[i] = ceil((double)segment_rows_trans[i] / blocksize);
            
            if(matrix_vector_mul == 0)
                gridsize_bwproj[i] = ceil(double(segment_rows_trans[i] + segment_nnzs_trans[i]) / (blocksize * secsize_bp));
            else
                gridsize_bwproj[i] = ceil((double)segment_rows_trans[i] / 32);        
        }   
        else{
            gridsize_update[i] = ceil((double)cols / blocksize);
            gridsize_bwproj[i] = ceil((double)segment_rows[i] / 32);
        }
        
    }

    // NCCL initialization
    ncclCommInitAll(comms, device_numbers, devices);

    // MLEM iterations
    for(unsigned int iter = 0; iter < iterations; iter++){
        
        // forward projection
        for(unsigned int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            if(matrix_vector_mul == 0)
                calcFwProj_merge_based <<< gridsize_fwproj[i], blocksize >>> (  
                    cuda_Rows[i], 
                    cuda_Cols[i], 
                    cuda_Vals[i], 
                    cuda_f[i],
                    using_trans == 0? cuda_temp[i] + segments[i] : cuda_temp[i], 
                    secsize_fp, 
                    segment_rows[i], 
                    segment_nnzs[i]);
            else
                calcFwProj_coalesced_brutal_warp <<< gridsize_fwproj[i], blocksize >>> (
                    cuda_Rows[i], 
                    cuda_Cols[i], 
                    cuda_Vals[i], 
                    cuda_f[i],
                    using_trans == 0 ? cuda_temp[i] + segments[i] : cuda_temp[i], 
                    segment_rows[i]);
        }

        // correlation
        for(unsigned int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            calcCorrel <<< gridsize_correl[i], blocksize >>> (
                cuda_g[i], 
                using_trans == 0 ? cuda_temp[i] + segments[i] : cuda_temp[i], 
                segment_rows[i]);
        }

        if(using_trans == 0){
            // sum up cuda_temp over devices
            ncclGroupStart();
            for (unsigned int i = 0; i < device_numbers; i++)
                ncclAllReduce((const void*)cuda_temp[i], (void*)cuda_temp[i], rows, ncclFloat, ncclSum, comms[i], streams[i]);
            ncclGroupEnd();
            for (unsigned int i = 0; i < device_numbers; i++) {
                hipSetDevice(i);
                hipStreamSynchronize(streams[i]);
            }
        }

        // backward projection
        for(unsigned int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            if(using_trans == 0){
                if(matrix_vector_mul == 0)
                    calcBwProj_merge_based <<< gridsize_bwproj[i], blocksize >>> (  
                        cuda_Rows_Trans[i], 
                        cuda_Cols_Trans[i], 
                        cuda_Vals_Trans[i], 
                        cuda_temp[i], 
                        cuda_bwproj[i] + segments_trans[i], 
                        secsize_bp, 
                        segment_rows_trans[i], 
                        segment_nnzs_trans[i]);
                else
                    calcBwProj_coalesced_brutal_warp <<< gridsize_bwproj[i], blocksize >>> (
                        cuda_Rows_Trans[i], 
                        cuda_Cols_Trans[i], 
                        cuda_Vals_Trans[i], 
                        cuda_temp[i], 
                        cuda_bwproj[i] + segments_trans[i], 
                        segment_rows_trans[i]);
            }
            else 
                calcBwProj_none_trans <<< gridsize_bwproj[i], blocksize >>> (
                    cuda_Rows[i], 
                    cuda_Cols[i], 
                    cuda_Vals[i],
                    cuda_temp[i], 
                    cuda_bwproj[i], 
                    segment_rows[i]);
        }

        // update
        for(unsigned int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            if(using_trans == 0)
                calcUpdate <<< gridsize_update[i], blocksize >>> (
                    cuda_f[i] + segments_trans[i], 
                    cuda_norm[i], 
                    cuda_bwproj[i] + segments_trans[i], 
                    segment_rows_trans[i]);
            else
                calcUpdate <<< gridsize_update[i], blocksize >>> (
                    cuda_f[i], 
                    cuda_norm[i], 
                    cuda_bwproj[i], 
                    cols);
        }

        // sum up cuda_bwproj over devices and save in cuda_f
        ncclGroupStart();
        for (unsigned int i = 0; i < device_numbers; i++)
            ncclAllReduce((const void*)cuda_bwproj[i], (void*)cuda_f[i], cols, ncclFloat, ncclSum, comms[i], streams[i]);
        ncclGroupEnd();
        for (unsigned int i = 0; i < device_numbers; i++) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }

        // clear cuda_bwproj
        for(unsigned int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            hipMemset(cuda_bwproj[i], 0, sizeof(float)*cols);
        }

        // clear cuda_temp
        for(unsigned int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            if(using_trans == 0)
                hipMemset(cuda_temp[i], 0, sizeof(float)*rows);
            else
                hipMemset(cuda_temp[i], 0, sizeof(float)*segment_rows[i]);
        }
    }

    // synchronize GPUs
    for (unsigned int i = 0; i < device_numbers; i++) {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }

    // Result is copied to f from device 0, actually now all devices hold the same result
    hipSetDevice(0);
    hipMemcpy(result_f, cuda_f[0], sizeof(float)*cols, hipMemcpyDeviceToHost);
    float sum = 0;
    for(unsigned int i = 0; i < cols; i++)
        sum += result_f[i];
    printf("\nSum f: %f\n", sum);
    

    // free all memory
    for(unsigned int i = 0; i < device_numbers; i++){
        hipSetDevice(i);
        ncclCommDestroy(comms[i]);
        if(cuda_Rows[i]) hipFree(cuda_Rows[i]);
        if(cuda_Cols[i]) hipFree(cuda_Cols[i]);
        if(cuda_Vals[i]) hipFree(cuda_Vals[i]);
        if(using_trans == 0){
            if(cuda_Rows_Trans[i]) hipFree(cuda_Rows_Trans[i]);
            if(cuda_Cols_Trans[i]) hipFree(cuda_Cols_Trans[i]);
            if(cuda_Vals_Trans[i]) hipFree(cuda_Vals_Trans[i]);
        }
        if(cuda_g[i]) hipFree(cuda_g[i]);
        if(cuda_norm[i]) hipFree(cuda_norm[i]);
        if(cuda_bwproj[i]) hipFree(cuda_bwproj[i]);
        if(cuda_temp[i]) hipFree(cuda_temp[i]);
        if(cuda_f[i]) hipFree(cuda_f[i]);
    }
    if(segments) free(segments);
    if(segment_rows) free(segment_rows);
    if(segment_nnzs) free(segment_nnzs);
    if(offsets) free(offsets);
    if(using_trans == 0){
        if(segments_trans) free(segments_trans);
        if(segment_rows_trans) free(segment_rows_trans);
        if(segment_nnzs_trans) free(segment_nnzs_trans);
        if(offsets_trans) free(offsets_trans);
    }
    if(comms) free(comms);
    if(streams) free(streams);
    if(devices) free(devices);
    if(cuda_Rows) free(cuda_Rows);
    if(cuda_Cols) free(cuda_Cols);
    if(cuda_Vals) free(cuda_Vals);
    if(using_trans == 0){
        if(cuda_Rows_Trans) free(cuda_Rows_Trans);
        if(cuda_Cols_Trans) free(cuda_Cols_Trans);
        if(cuda_Vals_Trans) free(cuda_Vals_Trans);
    }
    if(cuda_g) free(cuda_g);
    if(cuda_norm) free(cuda_norm);
    if(cuda_bwproj) free(cuda_bwproj);
    if(cuda_temp) free(cuda_temp);
    if(cuda_f) free(cuda_f);
    if(gridsize_fwproj) free(gridsize_fwproj);
    if(gridsize_correl) free(gridsize_correl);
    if(gridsize_bwproj) free(gridsize_bwproj);
    if(gridsize_update) free(gridsize_update);
}


void csr_format_for_cuda(const Csr4Matrix& matrix, float* csrVal, unsigned long* csrRowInd, unsigned int* csrColInd){   
    unsigned int index = 0;
    csrRowInd[index] = 0;
    unsigned int* tempIdx;
    tempIdx = (unsigned int*) malloc(sizeof(unsigned int) * matrix.rows());
    tempIdx[0] = 0;
    // !!! using openMP here will 100% lead to error in matrix
    // #pragma omp parallel for schedule (static)
    for (unsigned int row = 0; row < matrix.rows(); ++row) {
        csrRowInd[row + 1] = csrRowInd[row] + matrix.elementsInRow(row);
        index += matrix.elementsInRow(row);
        tempIdx[row + 1] = index;
    }

    #pragma omp parallel for 
    for (unsigned int row = 0; row < matrix.rows(); ++row) {
            /*
             auto it = matrix.beginRow2(row);
             unsigned int count = 0;
             unsigned int localindex = index;
             #pragma omp parallel for reduction(+:count)
             for(unsigned int i=0; i< (matrix.endRow2(row) - it); i++){
                csrVal[localindex + i] = (it+i)->value();
                csrColInd[localindex + i] = (unsigned int)((it+i)->column());
                count++;
            }
            index += count;*/
            unsigned int idx=0;
            std::for_each(matrix.beginRow2(row), matrix.endRow2(row),[&](const RowElement<float>& e){ 
                csrVal[tempIdx[row]+idx] = e.value();
                csrColInd[tempIdx[row]+idx] = (unsigned int)e.column() ;
                idx++;
            }
               // index = index + 1; }
            );
    }
}

void calcColumnSums(const Csr4Matrix& matrix, Vector<float>& norm)
{
    assert(matrix.columns() == norm.size());

    std::fill(norm.ptr(), norm.ptr() + norm.size(), 0.0);
    matrix.mapRows(0, matrix.rows());

    #pragma omp declare reduction(vec_float_plus : std::vector<float> : \
        std::transform(omp_out.begin(), omp_out.end(), omp_in.begin(), omp_out.begin(), std::plus<float>())) \
        initializer(omp_priv = omp_orig)
    
    std::vector<float> res(norm.size(),0);
    #pragma omp parallel for ordered reduction(vec_float_plus:res)
    for (uint32_t row=0; row<matrix.rows(); ++row) {
        std::for_each(matrix.beginRow2(row), matrix.endRow2(row),
                      [&](const RowElement<float>& e){ res[e.column()] += e.value(); });
    }
    #pragma omp parallel for 
    for(unsigned int i=0; i<norm.size(); i++){
        norm[i] = res[i];
    }

    // norm.writeToFile("norm-0.out");
}

/* a general version of halfMatrix: partition matrix unsigned into device_numbers parts, corresponding rows are saved in the array segments
   start row of segment i: segments[i]
    end  row of segment i: segments[i+1]
    number of rows in segment i: segments[i+1] - segments[i] (saved in segment_rows)
    number of nnzs in segment i: csr_Rows[segments[i+1]] - csr_Rows[segments[i]] (saved in segment_nnzs)
    offset when copying from host to device: csr_Rows[segments[i]] (saved in offsets)
*/
void partitionMatrix(unsigned long *csr_Rows, unsigned long nnzs, unsigned int rows, unsigned int device_numbers, unsigned int *segments, unsigned int *segment_rows, unsigned int *segment_nnzs, unsigned long *offsets){
    segments[0] = 0;
    segments[device_numbers] = rows;
    unsigned int i = 0;
    double nnzs_per_segment = ((double)nnzs / (double)device_numbers);
    for(unsigned int segment = 0; segment < device_numbers; segment++){
        for(; i <= rows; i++){
            if(csr_Rows[i] > nnzs_per_segment * segment){
                // printf("DEBUG: csr_Rows %lu > nnzs_per_segments %f * segments %u\n",  csr_Rows[i], nnzs_per_segment, segment);
                break;
            }
        }
        segments[segment] = i;
    }
    for(unsigned int segment = 0; segment < device_numbers; segment++){
        segment_rows[segment] = segments[segment+1] - segments[segment];
        segment_nnzs[segment] = (unsigned int)(csr_Rows[segments[segment+1]] - csr_Rows[segments[segment]]);
        offsets[segment] = csr_Rows[segments[segment]];
        // printf("Segment %u with rows: %u nnzs %u offset %lu\n", segment, segment_rows[segment], segment_nnzs[segment], offsets[segment]);
    }
}