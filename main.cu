#include "hip/hip_runtime.h"
#include "algorithm"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "math.h"
#include "kernel.cuh"
#include "hipsparse.h"
#include "csr4matrix.hpp"
#include "vector.hpp"
#include "time.h"
#include "sptrans.h"
#include "nccl.h"

void csr_format_for_cuda(const Csr4Matrix& matrix, float* csrVal, int* csrRowInd, int* csrColInd){   
    int index = 0;
    csrRowInd[index] = 0;
    // !!! using openMP here will 100% lead to error in matrix
    // #pragma omp parallel for schedule (static)
    for (int row = 0; row < matrix.rows(); ++row) {
        csrRowInd[row + 1] = csrRowInd[row] + (int)matrix.elementsInRow(row);
	
        std::for_each(matrix.beginRow2(row), matrix.endRow2(row),[&](const RowElement<float>& e){ 
            csrVal[index] = e.value();
            csrColInd[index] = (int)e.column() ;
            index = index + 1; }
        );
    }
}

void calcColumnSums(const Csr4Matrix& matrix, Vector<float>& norm)
{
    assert(matrix.columns() == norm.size());

    std::fill(norm.ptr(), norm.ptr() + norm.size(), 0.0);
    matrix.mapRows(0, matrix.rows());

  // pragma omp parallel for schedule (static)
    for (uint32_t row=0; row<matrix.rows(); ++row) {
        std::for_each(matrix.beginRow2(row), matrix.endRow2(row),
                      [&](const RowElement<float>& e){ norm[e.column()] += e.value(); });
    }
    // norm.writeToFile("norm-0.out");
}

void transposeCSR(int *cuda_Rows, int *cuda_Cols, float *cuda_Vals, int *cuda_Rows_Trans, int *cuda_Cols_Trans, float *cuda_Vals_Trans,
                    int rows, int cols, int nnzs){
    hipsparseStatus_t status;
	hipsparseHandle_t handle = 0;
	status = hipsparseCreate(&handle);
	if (status != HIPSPARSE_STATUS_SUCCESS){
        hipError_t cuda_err = hipGetLastError();
        printf("    Fail : CSR to CSC, cusparese initialization failed , ERROR %d, %s\n", status, hipGetErrorString(cuda_err));
    }
    status = hipsparseScsr2csc(handle, rows, cols, nnzs, cuda_Vals, cuda_Rows, cuda_Cols, cuda_Vals_Trans, cuda_Cols_Trans, cuda_Rows_Trans, 
                                HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
    if (status != HIPSPARSE_STATUS_SUCCESS)
        printf("    Fail : CSR to CSC, cusparse transpose failed\n");

    status = hipsparseDestroy(handle);
    handle = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS)
        printf("    Fail : CSR to CSC, cusparse destroy failed\n");
    
    // cusparse functions are asynchronous
    hipDeviceSynchronize();
}


// return row index for in which row the nnzs are distributed into two pars equally
int halfMatrix(int *csr_Rows, int nnzs, int rows){
    int i = 0;
    int halfnnzs = nnzs / 2;
    for(; i <= rows; i++)
        if(csr_Rows[i] >= halfnnzs)
            break;
    return i;
}

// return row index for in which row the nnzs are distributed into two pars equally
int fiveSixth(int *csr_Rows, int nnzs, int rows){
    int i = 0;
    double halfnnzs = (double)nnzs * 5.0 / 6.0;
    for(; i <= rows; i++)
        if(csr_Rows[i] >= halfnnzs)
            break;
    
    return i;
}

/* a general version of halfMatrix: partition matrix into device_numbers parts, corresponding rows are saved in the array segments
   start row of segment i: segments[i]
    end  row of segment i: segments[i+1]
    number of rows in segment i: segments[i+1] - segments[i] (saved in segment_rows)
    number of nnzs in segment i: csr_Rows[segments[i+1]] - csr_Rows[segments[i]] (saved in segment_nnzs)
    offset when copying from host to device: csr_Rows[segments[i]] (saved in offsets)
*/
void partitionMatrix(int *csr_Rows, int nnzs, int rows, int device_numbers, int *segments, int *segment_rows, int *segment_nnzs, int *offsets){
    segments[0] = 0;
    segments[device_numbers] = rows;
    int i = 0;
    int nnzs_per_segment = nnzs / device_numbers;
    for(int segment = 1; segment < device_numbers; segment++){
        for(; i <= rows; i++)
            if(csr_Rows[i] >= nnzs_per_segment * segment)
                break;
        segments[segment] = i;
    }
    for(int segment = 0; segment < device_numbers; segment++){
        segment_rows[segment] = segments[segment+1] - segments[segment];
        segment_nnzs[segment] = csr_Rows[segments[segment+1]] - csr_Rows[segments[segment]];
        offsets[segment] = csr_Rows[segments[segment]];
    }
}

void mlem_nccl( int *csr_Rows, int *csr_Cols, float *csr_Vals,
                int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, 
                int *g, float *norm, float *f, int rows, int cols, int nnzs, int iterations){
    
    int device_numbers;
    hipGetDeviceCount(&device_numbers);
    if(device_numbers < 2){
        printf("    \nWarning! Number of capable GPUs less than 2!\n\n");
        return;
    }
    else
        printf("    \nRunning NCCL MLEM with %d CUDA devices\n\n", device_numbers);

    clock_t start = clock();
    printf("    Begin: Initialization\n");
    clock_t initStart = clock();

    // partition matrix
    int *segments = (int*)malloc((device_numbers+1)*sizeof(int));
    int *segment_rows = (int*)malloc(device_numbers*sizeof(int));
    int *segment_nnzs = (int*)malloc(device_numbers*sizeof(int));
    int *offsets = (int*)malloc(device_numbers*sizeof(int));
    partitionMatrix(csr_Rows, nnzs, rows, device_numbers, segments, segment_rows, segment_nnzs, offsets);


    // partition transposed matrix
    int *segments_trans = (int*)malloc((device_numbers+1)*sizeof(int));
    int *segment_rows_trans = (int*)malloc(device_numbers*sizeof(int));
    int *segment_nnzs_trans = (int*)malloc(device_numbers*sizeof(int));
    int *offsets_trans = (int*)malloc(device_numbers*sizeof(int));
    partitionMatrix(csr_Rows_Trans, nnzs, cols, device_numbers, segments_trans, segment_rows_trans, segment_nnzs_trans, offsets_trans);
    
    
    // NCCL elements
    ncclComm_t *comms = (ncclComm_t*)malloc(device_numbers * sizeof(ncclComm_t));;
    hipStream_t *streams = (hipStream_t*)malloc(device_numbers * sizeof(hipStream_t));
    int *devices = (int*)malloc(device_numbers * sizeof(int));    


    // device variables
    int **cuda_Rows = (int**)malloc(device_numbers*sizeof(int*));
    int **cuda_Cols = (int**)malloc(device_numbers*sizeof(int*)); 
    int **cuda_Rows_Trans = (int**)malloc(device_numbers*sizeof(int*));
    int **cuda_Cols_Trans = (int**)malloc(device_numbers*sizeof(int*));
    int **cuda_g = (int**)malloc(device_numbers*sizeof(int*));
    float **cuda_Vals = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_Vals_Trans = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_norm = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_bwproj = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_temp = (float**)malloc(device_numbers*sizeof(float*));
    float **cuda_f = (float**)malloc(device_numbers*sizeof(float*));


    // initialization
    int blocksize = 1024;   // unique blocksize for all kernel calls
    int *gridsize_fwproj = (int*)malloc(device_numbers*sizeof(int));
    int *gridsize_correl = (int*)malloc(device_numbers*sizeof(int));
    int *gridsize_bwproj = (int*)malloc(device_numbers*sizeof(int));
    int *gridsize_update = (int*)malloc(device_numbers*sizeof(int));
    int *secsize_fwproj = (int*)malloc(device_numbers*sizeof(int));
    int *secsize_bwproj = (int*)malloc(device_numbers*sizeof(int));
    for(int i = 0; i < device_numbers; i++){
        hipSetDevice(i);
        hipStreamCreate(streams+i);
        devices[i] = i;

        hipMalloc((void**)&cuda_Rows[i], sizeof(int)*(segment_rows[i] + 1));
        hipMalloc((void**)&cuda_Cols[i], sizeof(int)*segment_nnzs[i]);
        hipMalloc((void**)&cuda_Vals[i], sizeof(float)*segment_nnzs[i]);
        hipMalloc((void**)&cuda_Rows_Trans[i], sizeof(int)*(segment_rows_trans[i] + 1));
        hipMalloc((void**)&cuda_Cols_Trans[i], sizeof(int)*segment_nnzs_trans[i]);
        hipMalloc((void**)&cuda_Vals_Trans[i], sizeof(float)*segment_nnzs_trans[i]);
        hipMalloc((void**)&cuda_f[i], sizeof(float)*cols);
        hipMalloc((void**)&cuda_bwproj[i], sizeof(float)*cols);
        hipMalloc((void**)&cuda_temp[i], sizeof(float)*rows);
        hipMalloc((void**)&cuda_g[i], sizeof(int)*segment_rows[i]);
        hipMalloc((void**)&cuda_norm[i], sizeof(float)*segment_rows_trans[i]);

        
        // copy matrix from host to devices
        for(int j = segments[i]; j <= segments[i+1]; j++ )
            csr_Rows[j] -= offsets[i];
        hipMemcpy(cuda_Rows[i], csr_Rows+segments[i], sizeof(int)*(segment_rows[i] + 1), hipMemcpyHostToDevice);
        csr_Rows[segments[i+1]] += offsets[i];
        hipMemcpy(cuda_Cols[i], csr_Cols+offsets[i], sizeof(int)*segment_nnzs[i], hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals[i], csr_Vals+offsets[i], sizeof(float)*segment_nnzs[i], hipMemcpyHostToDevice);
        
        // copy transposed matrix from host to devices
        for(int j = segments_trans[i]; j <= segments_trans[i+1]; j++ )
            csr_Rows_Trans[j] -= offsets_trans[i];
        hipMemcpy(cuda_Rows_Trans[i], csr_Rows_Trans+segments_trans[i], sizeof(int)*(segment_rows_trans[i] + 1), hipMemcpyHostToDevice);
        csr_Rows_Trans[segments_trans[i+1]] += offsets_trans[i];
        hipMemcpy(cuda_Cols_Trans[i], csr_Cols_Trans+offsets_trans[i], sizeof(int)*segment_nnzs_trans[i], hipMemcpyHostToDevice);
        hipMemcpy(cuda_Vals_Trans[i], csr_Vals_Trans+offsets_trans[i], sizeof(float)*segment_nnzs_trans[i], hipMemcpyHostToDevice);
        
        // copy other vectors from host to devices
        hipMemcpy(cuda_g[i], g+segments[i], sizeof(int)*segment_rows[i], hipMemcpyHostToDevice);
        hipMemcpy(cuda_norm[i], norm+segments_trans[i], sizeof(float)*segment_rows_trans[i], hipMemcpyHostToDevice);
        hipMemset(cuda_bwproj[i], 0, sizeof(float)*cols);
        hipMemset(cuda_temp[i], 0, sizeof(float)*rows);
        hipMemcpy(cuda_f[i], f, sizeof(float)*cols, hipMemcpyHostToDevice);
        
        // determine grid size for each step when calling CUDA kernels
        gridsize_correl[i] = ceil((double)segment_rows[i] / blocksize);
        gridsize_update[i] = ceil((double)segment_rows_trans[i] / blocksize);
        int items_fwproj = segment_rows[i] + segment_nnzs[i];
        int items_bwproj = segment_rows_trans[i] + segment_nnzs_trans[i];
        gridsize_fwproj[i] = ceil(sqrt((double)items_fwproj / blocksize));
        gridsize_bwproj[i] = ceil(sqrt((double)items_bwproj / blocksize));
        // determine section size for foward projection and backward projection
        secsize_fwproj[i] = ceil((double)items_fwproj / (blocksize * gridsize_fwproj[i]));
        secsize_bwproj[i] = ceil((double)items_bwproj / (blocksize * gridsize_bwproj[i]));
    }

    // NCCL initialization
    ncclCommInitAll(comms, device_numbers, devices);
    
    clock_t initEnd = clock();
    printf("    End  : Initialization\n");
    double initTime = ((double) (initEnd - initStart)) / CLOCKS_PER_SEC;
    printf("    Elapsed time for initialization: %f\n\n", initTime);


    // iterations
    printf("    Begin: Iterations %d\n", iterations);
    clock_t iterStart = clock();
    for(int iter = 0; iter < iterations; iter++){
        
        // forward projection
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            calcFwProj <<< gridsize_fwproj[i], blocksize >>> (  cuda_Rows[i], cuda_Cols[i], cuda_Vals[i], cuda_f[i], 
                                                                cuda_temp[i] + segments[i], secsize_fwproj[i], segment_rows[i], segment_nnzs[i]);
        }

        // correlation
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            calcCorrel <<< gridsize_correl[i], blocksize >>> (cuda_g[i], cuda_temp[i]+segments[i], segment_rows[i]);
        }

        // sum up cuda_temp over devices
        for (int i = 0; i < device_numbers; ++i) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }
        ncclGroupStart();
        for (int i = 0; i < device_numbers; i++)
            ncclAllReduce((const void*)cuda_temp[i], (void*)cuda_temp[i], rows, ncclFloat, ncclSum, comms[i], streams[i]);
        ncclGroupEnd();
        for (int i = 0; i < device_numbers; ++i) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }

        // backward projection
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            calcBwProj <<< gridsize_bwproj[i], blocksize >>> (  cuda_Rows_Trans[i], cuda_Cols_Trans[i], cuda_Vals_Trans[i], cuda_temp[i], 
                                                                cuda_bwproj[i] + segments_trans[i], secsize_bwproj[i], segment_rows_trans[i], segment_nnzs_trans[i]);
        }

        // update, for mlem nccl calcUpdate should be used, followd by clearing bwproj using cudamemset
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            calcUpdate <<< gridsize_update[i], blocksize >>> (cuda_f[i] + segments_trans[i], cuda_norm[i], cuda_bwproj[i] + segments_trans[i], segment_rows_trans[i]);
        }

        // sum up cuda_bwproj over devices and save in cuda_f
        for (int i = 0; i < device_numbers; ++i) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }
        ncclGroupStart();
        for (int i = 0; i < device_numbers; i++)
            ncclAllReduce((const void*)cuda_bwproj[i], (void*)cuda_f[i], cols, ncclFloat, ncclSum, comms[i], streams[i]);
        ncclGroupEnd();
        for (int i = 0; i < device_numbers; ++i) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }

        // clear cuda_bwproj
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            hipMemset(cuda_bwproj[i], 0, sizeof(float)*cols);
        }

        // clear cuda_temp
        for(int i = 0; i < device_numbers; i++){
            hipSetDevice(i);
            hipMemset(cuda_temp[i], 0, sizeof(float)*rows);
        }
    }
    for (int i = 0; i < device_numbers; ++i) {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }
    clock_t iterEnd = clock();
    printf("    End  : Iterations %d\n", iterations);
    double itertime = ((double) (iterEnd - iterStart)) / CLOCKS_PER_SEC;
    printf("    Elapsed time for iterations: %f\n\n", itertime);


    // Result is copied to f from device 0, actually now all devices hold the same result
    hipSetDevice(0);
    hipMemcpy(f, cuda_f[0], sizeof(float)*cols, hipMemcpyDeviceToHost);

    // free all memory
    for(int i = 0; i < device_numbers; i++){
        hipSetDevice(i);
        ncclCommDestroy(comms[i]);
        if(cuda_Rows[i]) hipFree(cuda_Rows[i]);
        if(cuda_Cols[i]) hipFree(cuda_Cols[i]);
        if(cuda_Rows_Trans[i]) hipFree(cuda_Rows_Trans[i]);
        if(cuda_Cols_Trans[i]) hipFree(cuda_Cols_Trans[i]);
        if(cuda_g[i]) hipFree(cuda_g[i]);
        if(cuda_Vals[i]) hipFree(cuda_Vals[i]);
        if(cuda_Vals_Trans[i]) hipFree(cuda_Vals_Trans[i]);
        if(cuda_norm[i]) hipFree(cuda_norm[i]);
        if(cuda_bwproj[i]) hipFree(cuda_bwproj[i]);
        if(cuda_temp[i]) hipFree(cuda_temp[i]);
        if(cuda_f[i]) hipFree(cuda_f[i]);
    }
    if(segments) free(segments);
    if(segment_rows) free(segment_rows);
    if(segment_nnzs) free(segment_nnzs);
    if(offsets) free(offsets);
    if(segments_trans) free(segments_trans);
    if(segment_rows_trans) free(segment_rows_trans);
    if(segment_nnzs_trans) free(segment_nnzs_trans);
    if(offsets_trans) free(offsets_trans);
    if(comms) free(comms);
    if(streams) free(streams);
    if(devices) free(devices);
    if(cuda_Rows) free(cuda_Rows);
    if(cuda_Cols) free(cuda_Cols);
    if(cuda_Rows_Trans) free(cuda_Rows_Trans);
    if(cuda_Cols_Trans) free(cuda_Cols_Trans);
    if(cuda_g) free(cuda_g);
    if(cuda_Vals) free(cuda_Vals);
    if(cuda_Vals_Trans) free(cuda_Vals_Trans);
    if(cuda_norm) free(cuda_norm);
    if(cuda_bwproj) free(cuda_bwproj);
    if(cuda_temp) free(cuda_temp);
    if(cuda_f) free(cuda_f);
    if(gridsize_fwproj) free(gridsize_fwproj);
    if(gridsize_correl) free(gridsize_correl);
    if(gridsize_bwproj) free(gridsize_bwproj);
    if(gridsize_update) free(gridsize_update);
    if(secsize_fwproj) free(secsize_fwproj);
    if(secsize_bwproj) free(secsize_bwproj);
    

    clock_t end = clock();
    double totaltime = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("    Elapsed time totally       : %f\n\n", totaltime);
}


void mlem_test(     int *csr_Rows, int *csr_Cols, float *csr_Vals, 
                    int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, 
                    int *g, float *norm, float *f, int rows, int cols, int nnzs, int iterations, int device, int matrix_vector_mul){

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);	
    hipSetDevice(device);
    printf("    \nRunning test MLEM on CUDA device %d (%s)\n\n", device, prop.name);

    clock_t start = clock();
    printf("    Begin: Initialization\n");
    clock_t initStart = clock();

    // device variables
    int *cuda_Rows, *cuda_Cols, *cuda_Rows_Trans, *cuda_Cols_Trans, *cuda_g;
    float *cuda_Vals, *cuda_Vals_Trans, *cuda_norm, *cuda_bwproj, *cuda_temp, *cuda_f;

    // allocate device storage
    hipMalloc((void**)&cuda_Rows, sizeof(int)*(rows + 1));
    hipMalloc((void**)&cuda_Cols, sizeof(int)*nnzs);
    hipMalloc((void**)&cuda_Vals, sizeof(float)*nnzs);
    hipMalloc((void**)&cuda_Rows_Trans, sizeof(int)*(cols + 1));
    hipMalloc((void**)&cuda_Cols_Trans, sizeof(int)*nnzs);
    hipMalloc((void**)&cuda_Vals_Trans, sizeof(float)*nnzs);
    hipMalloc((void**)&cuda_f, sizeof(float)*cols);
    hipMalloc((void**)&cuda_g, sizeof(int)*rows);
    hipMalloc((void**)&cuda_norm, sizeof(float)*cols);
    hipMalloc((void**)&cuda_bwproj, sizeof(float)*cols);
    hipMalloc((void**)&cuda_temp, sizeof(float)*rows);

    // value initialization
    hipMemcpy(cuda_Rows, csr_Rows, sizeof(int)*(rows + 1), hipMemcpyHostToDevice);
    hipMemcpy(cuda_Cols, csr_Cols, sizeof(int)* nnzs, hipMemcpyHostToDevice);
    hipMemcpy(cuda_Vals, csr_Vals, sizeof(float)* nnzs, hipMemcpyHostToDevice);
    hipMemcpy(cuda_Rows_Trans, csr_Rows_Trans, sizeof(int)*(cols + 1), hipMemcpyHostToDevice);
    hipMemcpy(cuda_Cols_Trans, csr_Cols_Trans, sizeof(int)* nnzs, hipMemcpyHostToDevice);
    hipMemcpy(cuda_Vals_Trans, csr_Vals_Trans, sizeof(float)* nnzs, hipMemcpyHostToDevice);
    hipMemcpy(cuda_g, g, sizeof(int)* rows, hipMemcpyHostToDevice);
    hipMemcpy(cuda_norm, norm, sizeof(float)* cols, hipMemcpyHostToDevice);
    hipMemset(cuda_bwproj, 0, sizeof(float)*cols);
    hipMemset(cuda_temp, 0, sizeof(float)*rows);
    hipMemcpy(cuda_f, f, sizeof(float)* cols, hipMemcpyHostToDevice);
    
    clock_t initEnd = clock();
    printf("    End  : Initialization\n");
    double initTime = ((double) (initEnd - initStart)) / CLOCKS_PER_SEC;
    printf("    Elapsed time for initialization: %f\n\n", initTime);


    // Determine grid size and section size (block size is set to 1024 by default)
    int blocksize = 1024;
    int gridsize_correl = ceil((double)rows / blocksize);
    int gridsize_update = ceil((double)cols / blocksize);

    // iterations
    printf("    Begin: Iterations %d\n", iterations);
    clock_t iterStart = clock();
    
    switch(matrix_vector_mul){
        case 0: { // case 0: CSRMV
            int items_fwproj = rows + nnzs;
            int items_bwproj = cols + nnzs;
            int gridsize_fwproj = ceil(sqrt((double)items_fwproj / blocksize) * 60); 
            int gridsize_bwproj = ceil(sqrt((double)items_bwproj / blocksize) * 15);
            int secsize_fwproj = ceil((double)items_fwproj / (blocksize * gridsize_fwproj));
            int secsize_bwproj = ceil((double)items_bwproj / (blocksize * gridsize_bwproj));
            
            for(int i = 0; i < iterations; i++){
                calcFwProj <<< gridsize_fwproj, blocksize >>> (cuda_Rows, cuda_Cols, cuda_Vals, cuda_f, cuda_temp, secsize_fwproj, rows, nnzs);
                
                calcCorrel <<< gridsize_correl, blocksize >>> (cuda_g, cuda_temp, rows);
        
                calcBwProj <<< gridsize_bwproj, blocksize >>> (cuda_Rows_Trans, cuda_Cols_Trans, cuda_Vals_Trans, cuda_temp, cuda_bwproj, secsize_bwproj, cols, nnzs);
                
                calcUpdateInPlace <<< gridsize_update, blocksize >>> (cuda_f, cuda_norm, cuda_bwproj, cols);
        
                hipMemset(cuda_temp,   0, sizeof(float)*rows);
                hipMemset(cuda_bwproj, 0, sizeof(float)*cols);     
            }
        } break;

        case 1: { //case 1: brutal
            int gridsize_fwproj = gridsize_correl;
            int gridsize_bwproj = gridsize_update;
            for(int i = 0; i < iterations; i++){
                calcFwProj_brutal <<< gridsize_fwproj, blocksize >>> (cuda_Rows, cuda_Cols, cuda_Vals, cuda_f, cuda_temp, rows);
        
                calcCorrel <<< gridsize_correl, blocksize >>> (cuda_g, cuda_temp, rows);
        
                calcBwProj_brutal <<< gridsize_bwproj, blocksize >>> (cuda_Rows_Trans, cuda_Cols_Trans, cuda_Vals_Trans, cuda_temp, cuda_bwproj, cols);
        
                calcUpdateInPlace <<< gridsize_update, blocksize >>> (cuda_f, cuda_norm, cuda_bwproj, cols);
            } 
        } break;

        case 2: { // case 2: coalesced CSRMV
            int items_fwproj = rows + nnzs;
            int items_bwproj = cols + nnzs;
            int gridsize_fwproj = ceil((double)items_fwproj / blocksize); 
            int gridsize_bwproj = ceil((double)items_bwproj / blocksize);
            for(int i = 0; i < iterations; i++){
                calcFwProj_coalesced <<< gridsize_fwproj, blocksize >>> (cuda_Rows, cuda_Cols, cuda_Vals, cuda_f, cuda_temp, blocksize, rows, nnzs);
                
                calcCorrel <<< gridsize_correl, blocksize >>> (cuda_g, cuda_temp, rows);
        
                calcBwProj_coalesced <<< gridsize_bwproj, blocksize >>> (cuda_Rows_Trans, cuda_Cols_Trans, cuda_Vals_Trans, cuda_temp, cuda_bwproj, blocksize, cols, nnzs);
                
                calcUpdateInPlace <<< gridsize_update, blocksize >>> (cuda_f, cuda_norm, cuda_bwproj, cols);
        
                hipMemset(cuda_temp,   0, sizeof(float)*rows);
                hipMemset(cuda_bwproj, 0, sizeof(float)*cols);     
            }
        } break;

        case 3: { // case 3: coalesced brutal
            for(int i = 0; i < iterations; i++){
                calcFwProj_coalesced_brutal <<< rows, blocksize >>> (cuda_Rows, cuda_Cols, cuda_Vals, cuda_f, cuda_temp);
        
                calcCorrel <<< gridsize_correl, blocksize >>> (cuda_g, cuda_temp, rows);
        
                calcBwProj_coalesced_brutal <<< cols, blocksize >>> (cuda_Rows_Trans, cuda_Cols_Trans, cuda_Vals_Trans, cuda_temp, cuda_bwproj);
        
                calcUpdateInPlace <<< gridsize_update, blocksize >>> (cuda_f, cuda_norm, cuda_bwproj, cols);
            } 
        } break;

        default: break;
    }
        
    hipDeviceSynchronize();
    clock_t iterEnd = clock();
    printf("    End  : Iterations %d\n", iterations);
    double itertime = ((double) (iterEnd - iterStart)) / CLOCKS_PER_SEC;
    printf("    Elapsed time for iterations: %f\n\n", itertime);

    // Result is copied to f
    hipMemcpy(f, cuda_f, sizeof(float)*cols, hipMemcpyDeviceToHost);

    // free all memory
    if(cuda_Rows) hipFree(cuda_Rows);
    if(cuda_Cols) hipFree(cuda_Cols);
    if(cuda_Vals) hipFree(cuda_Vals);
    if(cuda_Rows_Trans) hipFree(cuda_Rows_Trans);
    if(cuda_Cols_Trans) hipFree(cuda_Cols_Trans);
    if(cuda_Vals_Trans) hipFree(cuda_Vals_Trans);
    if(cuda_g) hipFree(cuda_g);
    if(cuda_norm) hipFree(cuda_norm);
    if(cuda_f) hipFree(cuda_f);
    if(cuda_bwproj) hipFree(cuda_bwproj);
    if(cuda_temp) hipFree(cuda_temp);


    clock_t end = clock();
    double totaltime = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("    Elapsed time totally       : %f\n\n", totaltime);
}


int main(){
    int iterations = 300;
    // 0: test mlem    1: naive mlme    other ints: nccl mlem
    int MLEM_Version = 1;
    // 0: using small matrix   1: using big matrix
    int small = 0;
    // 0: Quadro P6000 1: Tesla K20c
    int device = 0;
    // 0: CSRMV    1: brutal   2: coalesced CSRMV   3: coalesced brutal 
    int matrix_vector_mul = 0;

    printf("\nIteration times: ");
    int result = scanf("%d", &iterations);
    printf("\nMLEM version (0: test version   1: nccl version): ");
    result = scanf("%d", &MLEM_Version);
    printf("\nUse which matrix? (0: small matrix   1: big matrix): ");
    result = scanf("%d", &small);
    if(MLEM_Version == 0 || MLEM_Version == 1){
        printf("\nUse which device? (0: Quadro P6000   1: Tesla K20c): ");
        result = scanf("%d", &device);
        printf("\nUse which kind of matrix-vector multiplication? (0: CSRMV   1: brutal   2: coalesced CSRMV   3: coalesced brutal): ");
        result = scanf("%d", &matrix_vector_mul);
    }
    printf("\n");

    // host variables
    int *csr_Rows, *csr_Cols, *csr_Rows_Trans, *csr_Cols_Trans, *g, rows, cols, nnzs, sum_g = 0;
    float *csr_Vals, *csr_Vals_Trans, *f, *norm, sum_norm = 0.0f;


    // read matrix
    printf("Begin: Read Matrix\n");
    std::string matrixPath = small == 0? "/scratch/pet/madpet2.p016.csr4.small" : "/scratch/pet/madpet2.p016.csr4";
    Csr4Matrix matrix(matrixPath);
    printf("End  : Read Matrix\n\n");
    printf("Begin: Create CSR Format for Matrix\n");
    clock_t start = clock();
    rows = matrix.rows();
    cols = matrix.columns();
    nnzs = matrix.elements();
    printf("    The matrix contains %d rows, %d cols, %d nnzs\n", rows, cols, nnzs);
    matrix.mapRows(0, rows);    
    csr_Rows = (int*)malloc(sizeof(int) * (rows + 1));
    csr_Cols = (int*)malloc(sizeof(int) * nnzs);
    csr_Vals = (float*)malloc(sizeof(float) * nnzs);
    csr_format_for_cuda(matrix, csr_Vals, csr_Rows, csr_Cols);
    Vector<float> norm_helper(cols, 0.0);
    calcColumnSums(matrix, norm_helper);
    norm = norm_helper.ptr();
    // TODO: calculate sum_norm using gpu
    for(int i = 0; i < cols; i++)
        sum_norm += norm[i];
    clock_t end = clock();
    printf("End  : Create CSR Format for Matrix\n");
    double elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time for creating CSR: %f\n\n", elapsed);
    

    // read image
    printf("Begin: Read Image\n");
    start = clock();
    std::string imagePath = small == 0? "/scratch/pet/Trues_Derenzo_GATE_rot_sm_200k.LMsino.small" : "/scratch/pet/Trues_Derenzo_GATE_rot_sm_200k.LMsino";
    Vector<int> image(imagePath);
    g = image.ptr();
    // TODO: calculate sum_g using gpu
    for(int i = 0; i < rows; i++)
        sum_g += g[i];
    end = clock();
    printf("End  : Read Image\n");
    elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time for reading image: %f\n\n", elapsed);

    // calculate initial value
    float init = sum_g / sum_norm;
    printf("Sum of norms: %f\n", sum_norm);
    printf("Sum of g    : %d\n", sum_g);
    printf("Initial f   : %f\n\n", init);
    f = (float*)malloc(sizeof(float)*cols);
    for(int i = 0; i < cols; i++)
        f[i] = init;
    


    // !!!!!!!!!!!!!!!!!!!!!!
    if(small != 0){
        rows = fiveSixth(csr_Rows, nnzs, rows);
        nnzs = csr_Rows[rows];
        printf("\nNow rows is %d, nnzs is %d\n", rows, nnzs);
    }

    // transpose matrix
    printf("Begin: Transpose Matrix\n");
    start = clock();
    // transpose matrix using GPU
    // transposeCSR(cuda_Rows, cuda_Cols, cuda_Vals, cuda_Rows_Trans, cuda_Cols_Trans, cuda_Vals_Trans, rows, cols, nnzs);
    
    // transpose matrix using CPU
    csr_Rows_Trans = (int*) calloc (cols+1,sizeof(int));
    csr_Cols_Trans = (int*) calloc (nnzs,sizeof(int));
    csr_Vals_Trans = (float*) calloc (nnzs,sizeof(float));
    sptrans_scanTrans<int, float>(rows, cols, nnzs, csr_Rows, csr_Cols, csr_Vals, csr_Cols_Trans, csr_Rows_Trans, csr_Vals_Trans);
    end = clock();
    printf("End  : Transpose Matrix\n");
    elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time for transposing matrix: %f\n\n", elapsed);

    
    // run mlem algorithm matrix
    printf("\n***********************************************\n");
    printf("Begin: Run MLEM for %d iterations\n", iterations);
    switch(MLEM_Version){
        case 0: mlem_test(csr_Rows, csr_Cols, csr_Vals, csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, g, norm, f, rows, cols, nnzs, iterations, device, matrix_vector_mul); break;
        case 1: mlem_nccl(csr_Rows, csr_Cols, csr_Vals, csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, g, norm, f, rows, cols, nnzs, iterations); break;
        default: break;
    }
    printf("End  : Run MLEM for %d iterations\n", iterations);
    printf("***********************************************\n");

    // sum up all elements in the solution f
    float sum = 0;
    for(int i = 0; i < cols; i++)
        sum += f[i];
    
    printf("\nSum f: %f\n\n", sum);
    
    if (csr_Rows) free(csr_Rows);
    if (csr_Cols) free(csr_Cols);
    if (csr_Vals) free(csr_Vals);
    if (csr_Rows_Trans) free(csr_Rows_Trans);
    if (csr_Cols_Trans) free(csr_Cols_Trans);
    if (csr_Vals_Trans) free(csr_Vals_Trans);
    // if (g) free(g);
    // if (norm) free(norm);
    if (f) free(f);

    return 0;
}
