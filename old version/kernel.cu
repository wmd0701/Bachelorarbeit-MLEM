#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"

/*
	brief: calculate forward projection, output saved in fwproj
	@param csr_Row:		row array
	@param csr_Val:		value array
	@param csr_Col:		column array
	@param f:			f array from last iteration
	@param fwproj:		output array
	@param secSize:		section size
	@param rows:			number of rows (equals to length of row array - 1)
	@param nnzs:			number of nnzs (equals to length of val/col array)
*/
__global__ void calcFwProj_merge_based(	int *csr_Rows, int *csr_Cols, float *csr_Vals, float *f, float *fwproj, 
										int secSize, int rows, int nnzs) {
	
	// !!!  gridsize x blocksize x sectionsize		 >= rows + nnzs
	// !!! (gridsize x blocksize - 1) x sectionsize  <  rows + nnzs
	
	SpMV_start(csr_Rows, csr_Cols, csr_Vals, f, fwproj, secSize, rows, nnzs);
}


/*
	brief: calculate correlation, output saved in fwproj in-place
	@param g:			measurement array
	@param fwproj:		result of forward projection / output array
	@param rows:			number of rows (equals to length of row array - 1)
*/
__global__ void calcCorrel(int *g, float *fwproj, int rows) {
	
	// !!! gridsize x blocksize >= rows
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < rows) 
		if(fwproj[index] != 0.0f)
			fwproj[index] =  g[index] / fwproj[index];
}


/*
	brief: calculate backward projection using transposed matrix, output saved in bwproj
	@param csr_Row:		row array of transposed matrix
	@param csr_Val:		value array of transposed matrix
	@param csr_Col:		column array of transposed matrix
	@param correl:		result of correlation calculation
	@param bwproj:		output array
	@param secSize:		section size
	@param cols:			number of rows of transposed matrix (columns of original matrix)
	@param nnzs:			number of nnzs (equals to length of val/col array)
*/
__global__ void calcBwProj_merge_based(	int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, float *correl, float *bwproj,
										int secSize, int cols, int nnzs){

	// !!!  gridsize x blocksize x sectionsize		>= cols + nnzs
	// !!! (gridsize x blocksize - 1) x sectionsize <  cols + nnzs
	
	SpMV_start(csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, correl, bwproj, secSize, cols, nnzs);
}


/*
	brief: calculate update, output saved in bwproj, for mlem nccl
	@param f:			input array
	@param norm:		norm array
	@param bwproj:		result of backward projection / output array
	@param cols:		number of columns of original matrix
*/
__global__ void calcUpdate(float *f, float *norm, float *bwproj, int cols) {
	
	// !!!gridsize x blocksize >= cols

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < cols) {
		if(norm[index] == 0.0f)
			bwproj[index] = f[index] * bwproj[index];
		else
			bwproj[index] = f[index] * bwproj[index] / norm[index];
	}
}


/*
	brief: calculate update, output saved in f in-place, for mlem naive
	@param f:			input array / output array
	@param norm:		norm array
	@param bwproj:		result of backward projection
	@param cols:		number of columns of original matrix
*/
__global__ void calcUpdateInPlace(float *f, float *norm, float *bwproj, int cols) {
	
	// !!!gridsize x blocksize >= cols

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < cols) {
		if(norm[index] == 0)
			f[index] = f[index] * bwproj[index];
		else
			f[index] = f[index] * bwproj[index] / norm[index];
		
		// bwproj[index] = 0.0f;
	}
}

__global__ void calcFwProj_coalesced (int *csr_Rows, int *csr_Cols, float *csr_Vals, float *f, float *fwproj, int secSize, int rows, int nnzs) {
	SpMV_start_coalesced(csr_Rows, csr_Cols, csr_Vals, f, fwproj, secSize, rows, nnzs);
}

__global__ void calcBwProj_coalesced (int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, float *correl, float *bwproj, int secSize, int cols, int nnzs){
	SpMV_start_coalesced(csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, correl, bwproj, secSize, cols, nnzs);
}


__global__ void calcFwProj_brutal (int *csr_Rows, int *csr_Cols, float *csr_Vals, float *f, float *fwproj, int rows){
	mat_vec_mul_brutal(csr_Rows, csr_Cols, csr_Vals, f, fwproj, rows);
}


__global__ void calcBwProj_brutal (int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, float *correl, float *bwproj, int cols){
	mat_vec_mul_brutal(csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, correl, bwproj, cols);	
}

__global__ void calcFwProj_coalesced_brutal_block (int *csr_Rows, int *csr_Cols, float *csr_Vals, float *f, float *fwproj){
	mat_vec_mul_coalesced_brutal_block (csr_Rows, csr_Cols, csr_Vals, f, fwproj);
}
__global__ void calcBwProj_coalesced_brutal_block (int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, float *correl, float *bwproj){
	mat_vec_mul_coalesced_brutal_block (csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, correl, bwproj);
}

__global__ void calcFwProj_coalesced_brutal_warp (int *csr_Rows, int *csr_Cols, float *csr_Vals, float *f, float *fwproj, int rows){
	mat_vec_mul_coalesced_brutal_warp (csr_Rows, csr_Cols, csr_Vals, f, fwproj, rows);
}

__global__ void calcBwProj_coalesced_brutal_warp(int *csr_Rows_Trans, int *csr_Cols_Trans, float *csr_Vals_Trans, float *correl, float *bwproj, int cols){
	mat_vec_mul_coalesced_brutal_warp (csr_Rows_Trans, csr_Cols_Trans, csr_Vals_Trans, correl, bwproj, cols);
}

__global__ void calcBwProj_none_trans(int *csr_Rows, int *csr_Cols, float *csr_Vals, float *correl, float *bwproj, int rows){
	trans_mat_vec_mul_warp(csr_Rows, csr_Cols, csr_Vals, correl, bwproj, rows);
}






/*
	brief: find start coordinate for each section and call SpMV_work
	@param csr_Row:		row array
	@param csr_Val:		value array
	@param csr_Col:		column array
	@param *x:			vector being multiplied
	@param *result:		result vector
	@param secSize:		section size
	@param rows:			number of rows (equals to length of row array - 1)
	@param nnzs:			number of nnzs (equals to length of val/col array)
*/
__device__ void SpMV_start(	int *csr_Rows, int *csr_Cols, float *csr_Vals, float *x, float *result,
							int secSize, int rows, int nnzs) {
	
	// !!!  gridsize x blocksize x sectionsize		 >= rows + nnzs
	// !!! (gridsize x blocksize - 1) x sectionsize  <  rows + nnzs

	int lefti = 0;
	int righti = rows;
	int nexti = righti / 2;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int start = index * secSize;
	int nextj = start - nexti;
	int i = 0, j = start;

	while (i != nexti) {
		i = nexti;
		j = nextj;

		// find the first coordinate (i, j) that r[i + 1] > j - 1
		if (csr_Rows[i + 1] > j - 1)
			righti = i;
		else
			lefti = i + 1;

		nexti = (lefti + righti) / 2;
		nextj = start - nexti;

		/*
			nexti = righti only happens when index of diagonal (start) is exactly number of rows + number of nnz,
			which should not happen in reality
			if (nexti = righti)
				break;
		*/
	}

	SpMV_work(csr_Rows, csr_Cols, csr_Vals, x, result, secSize, rows, nnzs, i, j);
}


/*
	brief: matrix-vector multiplication for each section
	@param i:			x-coordinate of start point
	@param j:			y-coordinate of start point
	other params:		same as SpMV_start
*/
__device__ void SpMV_work(	int *csr_Rows, int *csr_Cols, float *csr_Vals, float *x, float *result,
							int secSize, int rows, int nnzs, int i, int j) {
	int end = i + j + secSize;
	if (end > nnzs + rows)
		end = nnzs + rows;
	float rowTimesVector = 0.0f;
	while (i + j < end) {
		if (csr_Rows[i + 1] > j) {
			rowTimesVector += csr_Vals[j] * x[csr_Cols[j]];
			j++;
		}
		else {
			// result[i++] += rowTimesVector;
			atomicAdd(result + i, rowTimesVector);
			i++;
			rowTimesVector = 0.0f;
		}
	}
	if (rowTimesVector != 0.0f)
		// result[i] += rowTimesVector;
		atomicAdd(result + i, rowTimesVector);
}


__device__ void SpMV_start_coalesced(	int *csr_Rows, int *csr_Cols, float *csr_Vals, float *x, float *result,
										int secSize, int rows, int nnzs) {

	int lefti = 0;
	int righti = rows;
	int nexti = righti / 2;
	int index = blockIdx.x;
	int start = index * secSize;
	int nextj = start - nexti;
	int i = 0, j = start;

	while (i != nexti) {
		i = nexti;
		j = nextj;

		// find the first coordinate (i, j) that r[i + 1] > j - 1
		if (csr_Rows[i + 1] > j - 1)
			righti = i;
		else
			lefti = i + 1;

		nexti = (lefti + righti) / 2;
		nextj = start - nexti;
	}

	SpMV_work_coalesced(csr_Rows, csr_Cols, csr_Vals, x, result, rows, nnzs, i, j);
}


__device__ void SpMV_work_coalesced(	int *csr_Rows, int *csr_Cols, float *csr_Vals, float *x, float *result,
										int rows, int nnzs, int i, int j) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < nnzs + rows){
		while (i + j != index) {
			if (csr_Rows[i + 1] > j) 
				j++;
			else 
				i++;
		}
		if (csr_Rows[i + 1] > j)
			atomicAdd(result + i, csr_Vals[j] * x[csr_Cols[j]]);
	}
}



__device__ void mat_vec_mul_brutal(int *csr_Rows, int *csr_Cols, float *csr_Vals, float *x, float *result, int rows){
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < rows){
		int start = csr_Rows[index];
		int end   = csr_Rows[index+1];
		float sum = 0.0f; 
		for(int i = start ; i < end ; i++)
			sum += csr_Vals[i] * x[csr_Cols[i]];
		
		result[index] = sum;
	}	
}


__device__ void mat_vec_mul_coalesced_brutal_block(int *csr_Rows, int *csr_Cols, float *csr_Vals, float *x, float *result){
	__shared__ float values[1024];
	for(int i = 0 ; i < 1024 ; i++)
		values[i] = 0.0f;
	__syncthreads();

	int threadIndex = threadIdx.x;
	int blockIndex  = blockIdx.x;
	int dim         = blockDim.x; // 1024

	int start = csr_Rows[blockIndex];
	int end   = csr_Rows[blockIndex + 1];

	for(int i = start + threadIndex; i < end ; i += dim)
		values[threadIndex] += csr_Vals[i] * x[csr_Cols[i]];

	__syncthreads();
	
	if(threadIndex == 0){
		float sum = 0.0f;
	
		for(int i = 0 ; i < 1024 ; i++)
			sum += values[i];

		result[blockIndex] = sum;
	}
}


__device__ void mat_vec_mul_coalesced_brutal_warp (int *csr_Rows, int *csr_Cols, float *csr_Vals, float *x, float *result, int rows){
	__shared__ float values[1024];

	int WARP_SIZE = 32;

	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

	int thread_lane = threadIdx.x & (WARP_SIZE-1); // thread index within the warp

	int warp_id = thread_id / WARP_SIZE; // global warp index
	// total number of active warps
	int num_warps = (blockDim.x / WARP_SIZE) * gridDim.x;
	// one warp per row
	for (int row = warp_id; row < rows ; row += num_warps){
		int row_start = csr_Rows[row];
		int row_end = csr_Rows[row + 1];
		
		// compute running sum per thread
		values[threadIdx.x] = 0.0;
		
		for (int jj = row_start + thread_lane ; jj < row_end ; jj += WARP_SIZE)
			values[threadIdx.x] += csr_Vals[jj] * x[csr_Cols[jj]];

		// first thread writes the result
		if (thread_lane == 0){
			for (int i = 1 ; i < WARP_SIZE ; i++)
				values[threadIdx.x] += values[threadIdx.x + i];
			
			atomicAdd(result + row, values[threadIdx.x]);
		}

		__syncthreads();
	}
}


__device__ void trans_mat_vec_mul_warp(int *csr_Rows, int *csr_Cols, float *csr_Vals, float *x, float *result, int rows){

	int WARP_SIZE = 32;

	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

	int thread_lane = threadIdx.x & (WARP_SIZE-1); // thread index within the warp

	int warp_id = thread_id / WARP_SIZE; // global warp index
	// total number of active warps
	int num_warps = (blockDim.x / WARP_SIZE) * gridDim.x;
	for(int row = warp_id; row < rows ; row += num_warps){
		int row_start = csr_Rows[row];
		int row_end   = csr_Rows[row + 1];
		for (int i= row_start + thread_lane; i < row_end; i += WARP_SIZE)
			atomicAdd(&result[csr_Cols[i]], csr_Vals[i] * x[row]);
	}
}
